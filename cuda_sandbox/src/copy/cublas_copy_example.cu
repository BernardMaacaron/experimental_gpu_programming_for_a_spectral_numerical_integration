/*
 * Copyright 2020 NVIDIA Corporation.  All rights reserved.
 *
 * NOTICE TO LICENSEE:
 *
 * This source code and/or documentation ("Licensed Deliverables") are
 * subject to NVIDIA intellectual property rights under U.S. and
 * international Copyright laws.
 *
 * These Licensed Deliverables contained herein is PROPRIETARY and
 * CONFIDENTIAL to NVIDIA and is being provided under the terms and
 * conditions of a form of NVIDIA software license agreement by and
 * between NVIDIA and Licensee ("License Agreement") or electronically
 * accepted by Licensee.  Notwithstanding any terms or conditions to
 * the contrary in the License Agreement, reproduction or disclosure
 * of the Licensed Deliverables to any third party without the express
 * written consent of NVIDIA is prohibited.
 *
 * NOTWITHSTANDING ANY TERMS OR CONDITIONS TO THE CONTRARY IN THE
 * LICENSE AGREEMENT, NVIDIA MAKES NO REPRESENTATION ABOUT THE
 * SUITABILITY OF THESE LICENSED DELIVERABLES FOR ANY PURPOSE.  IT IS
 * PROVIDED "AS IS" WITHOUT EXPRESS OR IMPLIED WARRANTY OF ANY KIND.
 * NVIDIA DISCLAIMS ALL WARRANTIES WITH REGARD TO THESE LICENSED
 * DELIVERABLES, INCLUDING ALL IMPLIED WARRANTIES OF MERCHANTABILITY,
 * NONINFRINGEMENT, AND FITNESS FOR A PARTICULAR PURPOSE.
 * NOTWITHSTANDING ANY TERMS OR CONDITIONS TO THE CONTRARY IN THE
 * LICENSE AGREEMENT, IN NO EVENT SHALL NVIDIA BE LIABLE FOR ANY
 * SPECIAL, INDIRECT, INCIDENTAL, OR CONSEQUENTIAL DAMAGES, OR ANY
 * DAMAGES WHATSOEVER RESULTING FROM LOSS OF USE, DATA OR PROFITS,
 * WHETHER IN AN ACTION OF CONTRACT, NEGLIGENCE OR OTHER TORTIOUS
 * ACTION, ARISING OUT OF OR IN CONNECTION WITH THE USE OR PERFORMANCE
 * OF THESE LICENSED DELIVERABLES.
 *
 * U.S. Government End Users.  These Licensed Deliverables are a
 * "commercial item" as that term is defined at 48 C.F.R. 2.101 (OCT
 * 1995), consisting of "commercial computer software" and "commercial
 * computer software documentation" as such terms are used in 48
 * C.F.R. 12.212 (SEPT 1995) and is provided to the U.S. Government
 * only as a commercial end item.  Consistent with 48 C.F.R.12.212 and
 * 48 C.F.R. 227.7202-1 through 227.7202-4 (JUNE 1995), all
 * U.S. Government End Users acquire the Licensed Deliverables with
 * only those rights set forth herein.
 *
 * Any use of the Licensed Deliverables in individual and commercial
 * software must include, in the user documentation and internal
 * comments to the code, the above Disclaimer and U.S. Government End
 * Users Notice.
 */

#include <cstdio>
#include <cstdlib>
#include <vector>

#include <hipblas.h>
#include <hip/hip_runtime.h>

#include "utilities.h"

using data_type = double;

int main(int argc, char *argv[]) {
    hipblasHandle_t cublasH = NULL;
    hipStream_t stream = NULL;

    /*
     *   A = | 1.0 2.0 3.0 4.0 |
     *   B = | 0.0 0.0 0.0 0.0 |
     */

    std::vector<data_type> A;
    const unsigned int n = 5;
    for(int i=1; i<= n*n; i++)
        A.push_back(i);


    



    const unsigned int m = 3;
    std::vector<data_type> B(m*m, 0);

    const int incx = 1;
    const int incy = 1;

    data_type *d_A = nullptr;
    data_type *d_B = nullptr;

    printf("A\n");
    print_matrix(n, n, A.data(), n);
    printf("=====\n");

    printf("B\n");
    print_matrix(m, m, B.data(), m);
    printf("=====\n");

    /* step 1: create cublas handle, bind a stream */
    CUBLAS_CHECK(hipblasCreate(&cublasH));

    CUDA_CHECK(hipStreamCreateWithFlags(&stream, hipStreamNonBlocking));
    CUBLAS_CHECK(hipblasSetStream(cublasH, stream));

    /* step 2: copy data to device */
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_A), sizeof(data_type) * A.size()));
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_B), sizeof(data_type) * B.size()));

    CUDA_CHECK(hipMemcpyAsync(d_A, A.data(), sizeof(data_type) * A.size(), hipMemcpyHostToDevice,                               
                                stream));
    CUDA_CHECK(hipMemcpyAsync(d_B, B.data(), sizeof(data_type) * B.size(), hipMemcpyHostToDevice,
                               stream));

    /* step 3: compute */
    //CUBLAS_CHECK(hipblasDcopy(cublasH, A.size(), d_A + 12, incx, d_B, incy));
    size_t d_B_pitch = m*sizeof(double); // the memory the length, in bytes, of the padded row
    const unsigned int submatrix_left_corner_row = 2;
    const unsigned int submatrix_left_corner_col = 2;
    const unsigned int offset_of_submatrix = n*submatrix_left_corner_col + submatrix_left_corner_row;

    size_t d_A_pitch = n*sizeof(double); // the memory the length, in bytes, of the padded row
    CUDA_CHECK(hipMemcpy2D(d_B, d_B_pitch, d_A + offset_of_submatrix, d_A_pitch, d_B_pitch, m, hipMemcpyDeviceToDevice));

    /* step 4: copy data to host */
    CUDA_CHECK(hipMemcpyAsync(B.data(), d_B, sizeof(data_type) * B.size(), hipMemcpyDeviceToHost,
                               stream));

    CUDA_CHECK(hipStreamSynchronize(stream));

    /*
     *   B = | 1.0 2.0 3.0 4.0 |
     */

    printf("B\n");
    print_matrix(m, m, B.data(), m);
    printf("=====\n");

    /* free resources */
    CUDA_CHECK(hipFree(d_A));
    CUDA_CHECK(hipFree(d_B));

    CUBLAS_CHECK(hipblasDestroy(cublasH));

    CUDA_CHECK(hipStreamDestroy(stream));

    CUDA_CHECK(hipDeviceReset());

    return EXIT_SUCCESS;
}
