#include "hip/hip_runtime.h"
#include <cstdio>
#include <cstdlib>
#include <vector>

#include <fstream>
#include <cmath>

#include <hipblas.h>
// // #include <hip/hip_runtime.h>

#include "spectral_integration_utilities.h"
#include "chebyshev_differentiation.h"
#include "lie_algebra_utilities.h"
#include "utilities.h"  //  Utilities for error handling (must be after cublas or cusolver)
#include "globals.h"
#include "getCusolverErrorString.h"

#include <Eigen/Dense>
#include <unsupported/Eigen/KroneckerProduct>


// CUDA specific variables
const auto size_of_double = sizeof(double);
hipsolverHandle_t cusolverH = NULL;
hipblasHandle_t cublasH = NULL;


static const unsigned int number_of_Chebyshev_points = 16;

static const unsigned int quaternion_state_dimension = 4;
static const unsigned int position_dimension = 3;
static const unsigned int quaternion_problem_dimension = quaternion_state_dimension * (number_of_Chebyshev_points-1);

static const unsigned int lambda_dimension = 6;

static const unsigned int Qa_dimension = 9;


// Defining qe in the CPU and its GPU parameters
Eigen::Matrix<double, ne*na, 1> qe;
double* d_qe = nullptr;
int size_of_qe_in_bytes = ne * na * size_of_double;


//  Obtain the Chebyshev differentiation matrix
const Eigen::MatrixXd Dn = getDn<number_of_Chebyshev_points>();

//FORWARD INTEGRATION:
//  Extract D_NN from the differentiation matrix (for the spectral integration)
const Eigen::MatrixXd Dn_NN_F = Dn.block<number_of_Chebyshev_points-1, number_of_Chebyshev_points-1>(0, 0);
//  Extract D_IN (for the propagation of initial conditions)
const Eigen::MatrixXd Dn_IN_F = Dn.block<number_of_Chebyshev_points-1, 1>(0, number_of_Chebyshev_points-1);

//BACKWARD INTEGRATION:
//  Extract D_NN from the differentiation matrix (for the spectral integration)
const Eigen::MatrixXd Dn_NN_B = Dn.block<number_of_Chebyshev_points-1, number_of_Chebyshev_points-1>(1, 1);
//  Extract D_IN (for the propagation of initial conditions)
const Eigen::MatrixXd Dn_IN_B = Dn.block<number_of_Chebyshev_points-1, 1>(1, 0);


// Define the Chebyshev points on the unit circle
const auto Chebyshev_points = ComputeChebyshevPoints<number_of_Chebyshev_points>();
Eigen::MatrixXd Phi_stack = Eigen::MatrixXd::Zero(na*number_of_Chebyshev_points, na*ne);
double* d_Phi_stack = nullptr;
int size_of_Phi_stack_in_bytes = (na * number_of_Chebyshev_points) * (na * ne) * size_of_double;


// K_stack parameters for GPU
double* d_K_stack = nullptr;
int size_of_K_stack_in_bytes = na * number_of_Chebyshev_points * size_of_double;






// Function Definitions

__global__ void computeCMatrixKernel(const double* d_K_stack, const double* D_NN, double* C_NN) {

    int i = threadIdx.x;


    // Extract the curvature from the strain and compute A_at_chebyshev_point
    // Z_at_chebyshev_point <<      0, -K(0),  -K(1),  -K(2),
    //                             K(0),     0,   K(2),  -K(1),
    //                             K(1), -K(2),      0,   K(0),
    //                             K(2),  K(1),  -K(0),      0;

    // A_at_chebyshev_point = 0.5 * Z_at_chebyshev_point;

    #pragma region Compute_C_NN
    if (i < number_of_Chebyshev_points-1) {
        int row = 0;
        int col = 1;
        int row_index = row * (number_of_Chebyshev_points - 1) + i;
        int col_index = col * (number_of_Chebyshev_points - 1) + i;
        C_NN[row_index * quaternion_state_dimension + col_index] = D_NN[row_index * quaternion_state_dimension + col_index] + 0.5*d_K_stack[3*i+0];


        row = 0;
        col = 2;
        row_index = row * (number_of_Chebyshev_points - 1) + i;
        col_index = col * (number_of_Chebyshev_points - 1) + i;
        C_NN[row_index * quaternion_state_dimension + col_index] = D_NN[row_index * quaternion_state_dimension + col_index] + 0.5*d_K_stack[3*i+1];

        row = 0;
        col = 3;
        row_index = row * (number_of_Chebyshev_points - 1) + i;
        col_index = col * (number_of_Chebyshev_points - 1) + i;
        C_NN[row_index * quaternion_state_dimension + col_index] = D_NN[row_index * quaternion_state_dimension + col_index] + 0.5*d_K_stack[3*i+2];

        row = 1;
        col = 0;
        row_index = row * (number_of_Chebyshev_points - 1) + i;
        col_index = col * (number_of_Chebyshev_points - 1) + i;
        C_NN[row_index * quaternion_state_dimension + col_index] = D_NN[row_index * quaternion_state_dimension + col_index] - 0.5*d_K_stack[3*i+0];

        row = 1;
        col = 2;
        row_index = row * (number_of_Chebyshev_points - 1) + i;
        col_index = col * (number_of_Chebyshev_points - 1) + i;
        C_NN[row_index * quaternion_state_dimension + col_index] = D_NN[row_index * quaternion_state_dimension + col_index] - 0.5*d_K_stack[3*i+2];

        row = 1;
        col = 3;
        row_index = row * (number_of_Chebyshev_points - 1) + i;
        col_index = col * (number_of_Chebyshev_points - 1) + i;
        C_NN[row_index * quaternion_state_dimension + col_index] = D_NN[row_index * quaternion_state_dimension + col_index] + 0.5*d_K_stack[3*i+1];

        row = 2;
        col = 0;
        row_index = row * (number_of_Chebyshev_points - 1) + i;
        col_index = col * (number_of_Chebyshev_points - 1) + i;
        C_NN[row_index * quaternion_state_dimension + col_index] = D_NN[row_index * quaternion_state_dimension + col_index] - 0.5*d_K_stack[3*i+1];

        row = 2;
        col = 1;
        row_index = row * (number_of_Chebyshev_points - 1) + i;
        col_index = col * (number_of_Chebyshev_points - 1) + i;
        C_NN[row_index * quaternion_state_dimension + col_index] = D_NN[row_index * quaternion_state_dimension + col_index] + 0.5*d_K_stack[3*i+2];

        row = 2;
        col = 3;
        row_index = row * (number_of_Chebyshev_points - 1) + i;
        col_index = col * (number_of_Chebyshev_points - 1) + i;
        C_NN[row_index * quaternion_state_dimension + col_index] = D_NN[row_index * quaternion_state_dimension + col_index] - 0.5*d_K_stack[3*i+0];

        row = 3;
        col = 0;
        row_index = row * (number_of_Chebyshev_points - 1) + i;
        col_index = col * (number_of_Chebyshev_points - 1) + i;
        C_NN[row_index * quaternion_state_dimension + col_index] = D_NN[row_index * quaternion_state_dimension + col_index] - 0.5*d_K_stack[3*i+2];

        row = 3;
        col = 1;
        row_index = row * (number_of_Chebyshev_points - 1) + i;
        col_index = col * (number_of_Chebyshev_points - 1) + i;
        C_NN[row_index * quaternion_state_dimension + col_index] = D_NN[row_index * quaternion_state_dimension + col_index] - 0.5*d_K_stack[3*i+1];

        row = 3;
        col = 2;
        row_index = row * (number_of_Chebyshev_points - 1) + i;
        col_index = col * (number_of_Chebyshev_points - 1) + i;
        C_NN[row_index * quaternion_state_dimension + col_index] = D_NN[row_index * quaternion_state_dimension + col_index] + 0.5*d_K_stack[3*i+0];
    }
    #pragma endregion

    // for (unsigned int row = 0; row < quaternion_state_dimension; ++row) {
    //     for (unsigned int col = 0; col < quaternion_state_dimension; ++col) {
    //         int row_index = row * (number_of_Chebyshev_points - 1) + i;
    //         int col_index = col * (number_of_Chebyshev_points - 1) + i;
    //         C_NN[row_index * quaternion_state_dimension + col_index] = D_NN[row_index * quaternion_state_dimension + col_index] - A_at_chebyshev_point(row, col);
    //     }
    // }
}

/*
Eigen::MatrixXd computeCMatrix(const Eigen::VectorXd &t_qe, const Eigen::MatrixXd &D_NN)
{
    Eigen::MatrixXd C_NN = D_NN;

    // Compute the memory occupation 
    const auto size_of_t_qe_in_bytes = t_qe.size()*size_of_double;
    const auto size_of_D_NN_in_bytes = D_NN.size() * size_of_double;
    const auto size_of_C_NN_in_bytes = C_NN.size() * size_of_double;
    
    // Create Pointers
    double* d_t_qe;
    double* d_D_NN;
    double* d_C_NN;

    // Allocate the memory
    CUDA_CHECK(
        hipMalloc(reinterpret_cast<void **>(&d_t_qe), size_of_t_qe_in_bytes)
    );
    CUDA_CHECK(
        hipMalloc(reinterpret_cast<void **>&d_D_NN, size_of_D_NN_in_bytes)
    );
    CUDA_CHECK(
        hipMalloc(reinterpret_cast<void **>&d_C_NN, size_of_C_NN_in_bytes)
    );

    //  Copy the data
    CUDA_CHECK(
        hipMemcpy(d_t_qe, t_qe.data(), size_of_t_qe_in_bytes, hipMemcpyHostToDevice)
    );
    CUDA_CHECK(
        hipMemcpy(d_D_NN, D_NN.data(), size_of_D_NN_in_bytes, hipMemcpyHostToDevice)
    );
    CUDA_CHECK(
        hipMemcpy(d_C_NN, C_NN.data(), size_of_C_NN_in_bytes, hipMemcpyHostToDevice)
    );

    // Launch kernel with one block
    int threadsPerBlock = number_of_Chebyshev_points;
    computeCMatrixKernel<<<1, threadsPerBlock>>>(d_t_qe, d_D_NN, d_C_NN);

    // Copy result back 
    CUDA_CHECK(
        hipMemcpy(C_NN.data(), d_C_NN, size_of_C_NN_in_bytes, hipMemcpyDeviceToHost);
    );

    // Free the memory
    CUDA_CHECK(
        hipFree(t_qe)
    );
    CUDA_CHECK(
        hipFree(D_NN)
    );
    CUDA_CHECK(
        hipFree(C_NN)
    );

    return C_NN;
}
*/

// Used to build Q_stack
Eigen::VectorXd integrateQuaternions()
{
    #pragma region K_stack

    // Allocate memory on the device
    CUDA_CHECK(hipMalloc((void**)&d_Phi_stack, size_of_Phi_stack_in_bytes));
    CUDA_CHECK(hipMalloc((void**)&d_qe, size_of_qe_in_bytes));
    CUDA_CHECK(hipMalloc((void**)&d_K_stack, size_of_K_stack_in_bytes));

    // Copy input data from host to device
    CUDA_CHECK(hipMemcpy(d_Phi_stack, Phi_stack.data(), size_of_Phi_stack_in_bytes, hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_qe, qe.data(), size_of_qe_in_bytes, hipMemcpyHostToDevice));

    // Define stride values
    int ld_Phi_stack = na; // Leading dimension of matrix A
    int inc_qe = 1; // Increment for x
    int inc_K_stack = 1; // Increment for y
    int stride_Phi_stack = na * (na * ne); // Stride between matrices in A
    int stride_qe = 0; // Stride between vectors in x
    int stride_K_stack = na; // Stride between vectors in y

    // Perform gemvStridedBatched operation
    double alpha_cublas = 1.0; // Scalar alpha
    double beta_cublas = 0.0; // Scalar beta
    CUBLAS_CHECK(hipblasDgemvStridedBatched(cublasH, HIPBLAS_OP_N, na, na*ne, &alpha_cublas, d_Phi_stack, ld_Phi_stack, stride_Phi_stack,
                                            d_qe, inc_qe, stride_qe, &beta_cublas, d_K_stack, inc_K_stack, stride_K_stack, number_of_Chebyshev_points));
    
    #pragma endregion


    //  Now stack the matrices in the diagonal of bigger ones (as meny times as the state dimension)
    const Eigen::MatrixXd D_NN = Eigen::KroneckerProduct<Eigen::MatrixXd,Eigen::MatrixXd>(Eigen::MatrixXd::Identity(quaternion_state_dimension, quaternion_state_dimension), Dn_NN_F);
    const Eigen::MatrixXd D_IN = Eigen::KroneckerProduct<Eigen::MatrixXd,Eigen::MatrixXd>(Eigen::MatrixXd::Identity(quaternion_state_dimension, quaternion_state_dimension), Dn_IN_F);


    //Compute C_NN
    Eigen::MatrixXd C_NN = D_NN;
    
    // Compute the memory occupation 
    const auto size_of_D_NN_in_bytes = D_NN.size() * size_of_double;
    
    // Create Pointers
    double* d_D_NN = nullptr;
    double* d_C_NN = nullptr;

    // Allocate the memory
    CUDA_CHECK(
        hipMalloc(reinterpret_cast<void **>(&d_qe), size_of_qe_in_bytes)
    );
    CUDA_CHECK(
        hipMalloc(reinterpret_cast<void **>(&d_D_NN), size_of_D_NN_in_bytes)
    );
    CUDA_CHECK(
        hipMalloc(reinterpret_cast<void **>(&d_C_NN), size_of_D_NN_in_bytes) //Same size as D_NN 
    );

    //  Copy the data
    CUDA_CHECK(
        hipMemcpy(d_D_NN, D_NN.data(), size_of_D_NN_in_bytes, hipMemcpyHostToDevice)
    );
    CUDA_CHECK(
        hipMemcpy(d_C_NN, C_NN.data(), size_of_D_NN_in_bytes, hipMemcpyHostToDevice)
    );

    // Launch kernel with one block
    computeCMatrixKernel<<<1, number_of_Chebyshev_points-1>>>(d_K_stack, d_D_NN, d_C_NN);

    // Free the memory
    CUDA_CHECK(
        hipFree(d_qe)
    );
    CUDA_CHECK(
        hipFree(d_D_NN)
    );

    Eigen::MatrixXd q_init(4,1);
    q_init << 1, 0, 0, 0;

    Eigen::MatrixXd b = Eigen::MatrixXd::Zero(quaternion_problem_dimension,1);

    // Dimension definition
    const int rows_D_IN = D_IN.rows();
    const int cols_D_IN = D_IN.cols();
    const int ld_D_IN = rows_D_IN;  

    const int rows_q_init = q_init.rows();
    const int cols_q_init = q_init.cols();
    const int ld_q_init = rows_q_init;

    const int rows_b = b.rows();
    const int cols_b = b.cols();
    const int ld_b = rows_b;

    const int rows_res = b.rows();
    const int cols_res = b.cols();
    
    const int rows_C_NN = C_NN.rows();
    const int cols_C_NN = C_NN.cols();
    const int ld_C_NN = rows_C_NN;

    const int rows_Q_stack = rows_C_NN;
    const int cols_Q_stack = cols_b;

    // LU factorization variables
    int info = 0;
    int lwork = 0;

    // Create Pointers
    double* d_D_IN = nullptr;
    double* d_q_init = nullptr;
    double* d_b = nullptr;
    double* d_res = nullptr;
    double* d_Q_stack = nullptr;
    //double* d_C_NN = nullptr;
    double* d_work = nullptr;
    int* d_info = nullptr;

    // Compute the memory occupation 
    const auto size_of_D_IN_in_bytes = size_of_double * D_IN.size();
    const auto size_of_q_init_in_bytes = size_of_double * q_init.size();
    const auto size_of_b_in_bytes = size_of_double * b.size();
    const auto size_of_res_in_bytes = size_of_double * rows_res * cols_res;
    const auto size_of_Q_stack_in_bytes = size_of_double * rows_Q_stack * cols_Q_stack;

    // Allocate the memory
    CUDA_CHECK(
        hipMalloc(reinterpret_cast<void **>(&d_D_IN), size_of_D_IN_in_bytes)
    );
    CUDA_CHECK(
        hipMalloc(reinterpret_cast<void **>(&d_q_init), size_of_q_init_in_bytes)
    );
    CUDA_CHECK(
        hipMalloc(reinterpret_cast<void **>(&d_b), size_of_b_in_bytes)
    );
    CUDA_CHECK(
        hipMalloc(reinterpret_cast<void **>(&d_res), size_of_res_in_bytes)
    );
    CUDA_CHECK(
        hipMalloc(reinterpret_cast<void **>(&d_Q_stack), size_of_Q_stack_in_bytes)
    );
    // CUDA_CHECK(
    //     hipMalloc(reinterpret_cast<void **>(&d_C_NN), size_of_C_NN_in_bytes)
    // );
    CUDA_CHECK(
        hipMalloc(reinterpret_cast<void **>(&d_info), sizeof(int))
    );

    //  Copy the data: hipMemcpy(destination, file_to_copy, size_of_the_file, std_cmd)
    CUDA_CHECK(
        hipMemcpy(d_D_IN, D_IN.data(), size_of_D_IN_in_bytes, hipMemcpyHostToDevice)
    );
    CUDA_CHECK(
        hipMemcpy(d_q_init, q_init.data(), size_of_q_init_in_bytes, hipMemcpyHostToDevice)
    );
    CUDA_CHECK(
        hipMemcpy(d_b, b.data(), size_of_b_in_bytes, hipMemcpyHostToDevice)
    );
    // CUDA_CHECK(
    //     hipMemcpy(d_C_NN, C_NN.data(), size_of_C_NN_in_bytes, hipMemcpyHostToDevice)
    // );
    CUDA_CHECK(
        hipMemcpy(d_info, &info, sizeof(int), hipMemcpyHostToDevice)
    );

    // Allocates buffer size for the LU decomposition
    hipsolverStatus_t status = hipsolverDnDgetrf_bufferSize(cusolverH, rows_C_NN, cols_Q_stack, d_C_NN, ld_C_NN, &lwork);
    if (status != HIPSOLVER_STATUS_SUCCESS)
    {
        std::cerr << "cusolver error: " << getCusolverErrorString(status) << std::endl;
    };
    // Allocate the memory for LU factorization workspace
    CUDA_CHECK(
        hipMalloc(reinterpret_cast<void **>(&d_work), size_of_double * lwork)
    );

    //What we want to calculate
    Eigen::MatrixXd Q_stack_CUDA(rows_Q_stack, cols_Q_stack);

    // Computing b = -D_IN*q_init + b
    alpha_cublas = -1.0;
    beta_cublas = 1.0;


    CUBLAS_CHECK(
        hipblasDgemm(cublasH, HIPBLAS_OP_N, HIPBLAS_OP_N, rows_D_IN, cols_q_init, cols_D_IN, &alpha_cublas, d_D_IN, ld_D_IN, d_q_init, ld_q_init, &beta_cublas, d_b, ld_b)
    );

    // LU factorization
    CUSOLVER_CHECK(
        hipsolverDnDgetrf(cusolverH, rows_C_NN, cols_C_NN, d_C_NN, ld_C_NN, d_work, NULL, d_info)
    );

    // Solving the final system
    CUSOLVER_CHECK(
        hipsolverDnDgetrs(cusolverH, HIPBLAS_OP_N, rows_C_NN, 1, d_C_NN, ld_C_NN, NULL, d_b, ld_b, d_info)
    );

    // Memory Copy
    CUDA_CHECK(
        hipMemcpy(Q_stack_CUDA.data(), d_b, size_of_b_in_bytes, hipMemcpyDeviceToHost)
    );

    //FREEING MEMORY
    CUDA_CHECK(
        hipFree(d_D_IN)
    );
    CUDA_CHECK(
        hipFree(d_C_NN)
    );
    CUDA_CHECK(
        hipFree(d_b)
    );
    CUDA_CHECK(
        hipFree(d_info)
    );
    CUDA_CHECK(
        hipFree(d_q_init)
    );
    CUDA_CHECK(
        hipFree(d_Q_stack)
    );
    CUDA_CHECK(
        hipFree(d_res)
    );
    CUDA_CHECK(
        hipFree(d_work)
    );


    return Q_stack_CUDA;
}







// Used to build r_stack

__device__ void quaternionToRotationMatrix(const double* q, double* R) {
    double q0 = q[0];
    double q1 = q[1];
    double q2 = q[2];
    double q3 = q[3];

    R[0] = q0 * q0 + q1 * q1 - q2 * q2 - q3 * q3;
    R[1] = 2.0 * (q1 * q2 - q0 * q3);
    R[2] = 2.0 * (q1 * q3 + q0 * q2);
    R[3] = 2.0 * (q1 * q2 + q0 * q3);
    R[4] = q0 * q0 - q1 * q1 + q2 * q2 - q3 * q3;
    R[5] = 2.0 * (q2 * q3 - q0 * q1);
    R[6] = 2.0 * (q1 * q3 - q0 * q2);
    R[7] = 2.0 * (q2 * q3 + q0 * q1);
    R[8] = q0 * q0 - q1 * q1 - q2 * q2 + q3 * q3;
}

__global__ void updatePositionbKernel(double* t_Q_stack_CUDA, double* t_b){
    
    int i = threadIdx.x;

    Eigen::Map<Eigen::Matrix<double, Eigen::Dynamic, Eigen::Dynamic, Eigen::RowMajor>> b(t_b, number_of_Chebyshev_points-1, position_dimension);

    if (i < number_of_Chebyshev_points-1) {
        double q[4] = { t_Q_stack_CUDA[i], 
                        t_Q_stack_CUDA[i + (number_of_Chebyshev_points-1)],
                        t_Q_stack_CUDA[i + 2*(number_of_Chebyshev_points-1)], 
                        t_Q_stack_CUDA[i + 3*(number_of_Chebyshev_points-1)]
                        };

        double R[9];

        quaternionToRotationMatrix(q, R);

        b.block<1, 3>(i, 0) = (Eigen::Map<Eigen::MatrixXd>(R, 3, 3) * Eigen::Vector3d(1, 0, 0)).transpose();
    }
}

_global_ void computeIvpKernel(double* t_Dn_IN_F, double* t_r_init, double* t_ivp) {
    int i = threadIdx.x;

    if (i < number_of_Chebyshev_points-1) {
        t_ivp[i] = t_Dn_IN_F[i] * t_r_init[0];
        t_ivp[i+1] = t_Dn_IN_F[i] * t_r_init[1];
        t_ivp[i+2] = t_Dn_IN_F[i] * t_r_init[2];
    }
}

Eigen::MatrixXd integratePosition(Eigen::MatrixXd t_Q_stack_CUDA)
{   
    Eigen::Vector3d r_init;
    r_init << 0,
              0,
              0;

    //  This matrix remains constant so we can pre invert
    Eigen::MatrixXd Dn_NN_inv = Dn_NN_F.inverse();

    Eigen::Matrix<double, number_of_Chebyshev_points-1, position_dimension> b; //used in updatePositionbKernel
    
    Eigen::MatrixXd ivp(number_of_Chebyshev_points-1, position_dimension);

    Eigen::MatrixXd res(number_of_Chebyshev_points-1, position_dimension);

    // Compute the memory occupation
    const auto size_of_Q_stack_CUDA_in_bytes = t_Q_stack_CUDA.size() * size_of_double;
    const auto size_of_b_in_bytes = b.size() * size_of_double;
    const auto size_of_res_in_bytes = res.size() * size_of_double;
    const auto size_of_r_init_in_bytes = r_init.size() * size_of_double;
    const auto size_of_ivp_in_bytes = ivp.size() * size_of_double;
    const auto size_of_Dn_IN_F_in_bytes = Dn_IN_F.size() * size_of_double;

    // Create Pointers
    double* d_Q_stack_CUDA;
    double* d_b;
    double* d_r_init;
    double* d_ivp;
    double* d_Dn_IN_F;

    // Allocate the memory
    CUDA_CHECK(
        hipMalloc(reinterpret_cast<void **>(&d_Q_stack_CUDA), size_of_Q_stack_CUDA_in_bytes)
    );
    CUDA_CHECK(
        hipMalloc(reinterpret_cast<void **>(&d_b), size_of_b_in_bytes)
    );
    CUDA_CHECK(
        hipMalloc(reinterpret_cast<void **>(&d_r_init), size_of_r_init_in_bytes)
    );
    CUDA_CHECK(
        hipMalloc(reinterpret_cast<void **>(&d_ivp), size_of_ivp_in_bytes)
    );
    CUDA_CHECK(
        hipMalloc(reinterpret_cast<void **>(&d_Dn_IN_F), size_of_Dn_IN_F_in_bytes)
    );

    //  Copy the data
    CUDA_CHECK(
        hipMemcpy(d_Q_stack_CUDA, t_Q_stack_CUDA.data(), size_of_Q_stack_CUDA_in_bytes, hipMemcpyHostToDevice)
    );
    CUDA_CHECK(
        hipMemcpy(d_r_init, r_init.data(), size_of_r_init_in_bytes, hipMemcpyHostToDevice)
    );
    CUDA_CHECK(
        hipMemcpy(d_Dn_IN_F, Dn_IN_F.data(), size_of_Dn_IN_F_in_bytes, hipMemcpyHostToDevice)
    );

    // Launch the kernel for b: the result of the kernel is stored into d_b
    updatePositionbKernel<<<number_of_Chebyshev_points-1>>>(d_Q_stack_CUDA, d_b);
    
    // Launch the kernel for ivp: the result of the kernel is stored into d_b
    computeIvpKernel<<<number_of_Chebyshev_points-1>>>(d_Dn_IN_F, d_r_init, d_ivp);

    // Before we had b_NN = updatePositionb and thn res = B_NN -ivp so we have to do the same somehow 

    // Now the operation i have to perform is res = -d_ivp+d_b into the GPU

    // Dimensions
    const int ld_b = b.rows();
    const int rows_ivp = ivp.rows();
    const int cols_ivp = ivp.cols();
    const int ld_ivp = rows_ivp;

    // Computing b = -ivp + b       
    double alpha_cublas = -1.0;
    double beta_cublas = 1.0;

    CUBLAS_CHECK(
        hipblasDgemm(cublasH, HIPBLAS_OP_N, HIPBLAS_OP_N, rows_ivp, cols_ivp, cols_ivp, &alpha_cublas, d_ivp, ld_ivp, nullptr, ld_ivp, &beta_cublas, d_b, ld_b)
    );

    // This passage is for sure not necessary but right now it's the fastes thing. If everything work we will fix it.
    CUDA_CHECK(
        hipMemcpy(res.data(), d_b, size_of_res_in_bytes, hipMemcpyDeviceToHost)
    );

    // Define dimensions
    const int rows_Dn_NN_inv = Dn_NN_inv.rows();
    const int cols_Dn_NN_inv = Dn_NN_inv.cols();
    const int ld_Dn_NN_inv = rows_Dn_NN_inv;  

    const int rows_res = res.rows();
    const int cols_res = res.cols();
    const int ld_res = rows_res;

    const int rows_r_stack = rows_Dn_NN_inv;
    const int cols_r_stack = cols_res;
    const int ld_r_stack = rows_r_stack;

    // Create Pointers
    double* d_Dn_NN_inv = nullptr;
    double* d_res = nullptr;
    double* d_r_stack = nullptr;

    // Compute the memory occupation
    const auto size_of_Dn_NN_inv_in_bytes = size_of_double * Dn_NN_inv.size();
    const auto size_of_r_stack_in_bytes = size_of_double * rows_r_stack * cols_r_stack;

    // Allocate the memory
    CUDA_CHECK(
        hipMalloc(reinterpret_cast<void **>(&d_Dn_NN_inv), size_of_Dn_NN_inv_in_bytes)
    );
    CUDA_CHECK(
        hipMalloc(reinterpret_cast<void **>(&d_res), size_of_res_in_bytes)
    );
        CUDA_CHECK(
        hipMalloc(reinterpret_cast<void **>(&d_r_stack), size_of_r_stack_in_bytes)
    );
    
    //  Copy the data: hipMemcpy(destination, file_to_copy, size_of_the_file, std_cmd)
    CUDA_CHECK(
        hipMemcpy(d_Dn_NN_inv, Dn_NN_inv.data(), size_of_Dn_NN_inv_in_bytes, hipMemcpyHostToDevice)
    );
    CUDA_CHECK(
        hipMemcpy(d_res, res.data(), size_of_res_in_bytes, hipMemcpyHostToDevice)
    );

    // What we want to calculate 
    Eigen::MatrixXd r_stack_CUDA(rows_r_stack, cols_r_stack);

    // Compute r_stack = Dn_NN_inv*res
    alpha_cublas = 1.0;
    beta_cublas = 0.0;
    CUBLAS_CHECK(
        hipblasDgemm(cublasH, HIPBLAS_OP_N, HIPBLAS_OP_N, rows_Dn_NN_inv, cols_res, cols_Dn_NN_inv, &alpha_cublas, d_Dn_NN_inv, ld_Dn_NN_inv, d_res, ld_res, &beta_cublas, d_r_stack, ld_r_stack)
    );

    CUDA_CHECK(
        hipMemcpy(r_stack_CUDA.data(), d_r_stack, size_of_r_stack_in_bytes, hipMemcpyDeviceToHost)
    );

    //FREEING MEMORY
    CUDA_CHECK(
        hipFree(d_Dn_NN_inv)
    );
    CUDA_CHECK(
        hipFree(d_r_stack)
    );
    CUDA_CHECK(
        hipFree(d_res)
    );

    return r_stack_CUDA;
}








// Used to build Lambda_stack:
__global__ void updateCMatrixKernel(const double* d_K_stack, const double* D_NN, double* C_NN) {

    int i = threadIdx.x;

    // // Transpose of the K_hat matrix is equivalent to the K_hat matrix itself multiplied by -1

    // v_hat <<  0   ,  -t_v(2),   t_v(1),
    //         t_v(2),     0   ,  -t_v(0),
    //        -t_v(1),   t_v(0),     0   ;

    // v_hat TRANSPOSE <<  0   ,  t_v(2),   -t_v(1),
    //                  -t_v(2),     0   ,   t_v(0),
    //                  t_v(1),   -t_v(0),     0   ;


    // C_NN(row_index, col_index) = D_NN(row_index, col_index) - K_hat.transpose()(row, col) becomes
    // C_NN(row_index, col_index) = D_NN(row_index, col_index) + K_hat(row, col)

    // for (unsigned int row = 0; row < lambda_dimension/2; ++row) {
    //     for (unsigned int col = 0; col < lambda_dimension/2; ++col) {
    //         C_NN(row_index, col_index) = D_NN(row_index, col_index) + K_hat(row, col);
    //     }
    // }

    #pragma region Compute_C_NN
    if (i < number_of_Chebyshev_points-1) {
    int row = 0;
    int col = 1;
    int row_index = row * (number_of_Chebyshev_points - 1) + i;
    int col_index = col * (number_of_Chebyshev_points - 1) + i;
    C_NN[row_index * quaternion_state_dimension + col_index] = D_NN[row_index * quaternion_state_dimension + col_index] - d_K_stack[3*i+2];


    row = 0;
    col = 2;
    row_index = row * (number_of_Chebyshev_points - 1) + i;
    col_index = col * (number_of_Chebyshev_points - 1) + i;
    C_NN[row_index * quaternion_state_dimension + col_index] = D_NN[row_index * quaternion_state_dimension + col_index] + d_K_stack[3*i+1];


    row = 1;
    col = 0;
    row_index = row * (number_of_Chebyshev_points - 1) + i;
    col_index = col * (number_of_Chebyshev_points - 1) + i;
    C_NN[row_index * quaternion_state_dimension + col_index] = D_NN[row_index * quaternion_state_dimension + col_index] + d_K_stack[3*i+2];

    row = 1;
    col = 2;
    row_index = row * (number_of_Chebyshev_points - 1) + i;
    col_index = col * (number_of_Chebyshev_points - 1) + i;
    C_NN[row_index * quaternion_state_dimension + col_index] = D_NN[row_index * quaternion_state_dimension + col_index] - d_K_stack[3*i+0];

    row = 2;
    col = 0;
    row_index = row * (number_of_Chebyshev_points - 1) + i;
    col_index = col * (number_of_Chebyshev_points - 1) + i;
    C_NN[row_index * quaternion_state_dimension + col_index] = D_NN[row_index * quaternion_state_dimension + col_index] - d_K_stack[3*i+1];

    row = 2;
    col = 1;
    row_index = row * (number_of_Chebyshev_points - 1) + i;
    col_index = col * (number_of_Chebyshev_points - 1) + i;
    C_NN[row_index * quaternion_state_dimension + col_index] = D_NN[row_index * quaternion_state_dimension + col_index] + d_K_stack[3*i+0];
    }
    #pragma endregion
}

__global__ void computeNbarKernel(const double* t_Q_stack_CUDA_data, double* Nbar_stack_data) {
int i = threadIdx.x;

    const double g = 9.81; // m/s^2
    const double radius = 0.001; // m
    const double A = M_PI * radius * radius;
    const double rho = 7800; // kg/m^3

    Eigen::Map<const Eigen::VectorXd> t_Q_stack_CUDA(t_Q_stack_CUDA_data, (number_of_Chebyshev_points - 1) * 4);
    Eigen::Map<Eigen::VectorXd> Nbar_stack(Nbar_stack_data, (number_of_Chebyshev_points - 1) * 3);

    if (i < number_of_Chebyshev_points - 1) {
        Eigen::Quaterniond Qbar(t_Q_stack_CUDA(i), t_Q_stack_CUDA(i + (number_of_Chebyshev_points - 1)),
                                t_Q_stack_CUDA(i + 2 * (number_of_Chebyshev_points - 1)),
                                t_Q_stack_CUDA(i + 3 * (number_of_Chebyshev_points - 1)));

        double R[9];
        quaternionToRotationMatrix(Qbar.coeffs().data(), R); //Qbar.coeffs().data() returns a pointer to the raw data of the quaternion coefficients of Qbar

        double Fg = -A * g * rho;
        double Nbar[3] = {  R[6]*Fg, 
                            R[7]*Fg,
                            R[8]*Fg
        };

        Nbar_stack(i) = Nbar[0];
        Nbar_stack(i + (number_of_Chebyshev_points - 1)) = Nbar[1];
        Nbar_stack(i + 2 * (number_of_Chebyshev_points - 1)) = Nbar[2];
    }
}

Eigen::MatrixXd integrateInternalForces(Eigen::MatrixXd t_Q_stack_CUDA)
{   
    //  Now stack the matrices in the diagonal of bigger ones (as meny times as the state dimension)
    const Eigen::MatrixXd D_NN = Eigen::KroneckerProduct<Eigen::MatrixXd,Eigen::MatrixXd>(Eigen::MatrixXd::Identity(lambda_dimension/2, lambda_dimension/2), Dn_NN_B);
    const Eigen::MatrixXd D_IN = Eigen::KroneckerProduct<Eigen::MatrixXd,Eigen::MatrixXd>(Eigen::MatrixXd::Identity(lambda_dimension/2, lambda_dimension/2), Dn_IN_B);

    //Compute C_NN
    Eigen::MatrixXd C_NN = D_NN;
    
    // Compute the memory occupation 
    const auto size_of_D_NN_in_bytes = D_NN.size() * size_of_double;
    
    // Create Pointers
    double* d_D_NN = nullptr;
    double* d_C_NN = nullptr;

    // Allocate the memory
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_qe), size_of_qe_in_bytes));
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_D_NN), size_of_D_NN_in_bytes));
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_C_NN), size_of_D_NN_in_bytes));

    //  Copy the data
    CUDA_CHECK(hipMemcpy(d_D_NN, D_NN.data(), size_of_D_NN_in_bytes, hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_C_NN, C_NN.data(), size_of_D_NN_in_bytes, hipMemcpyHostToDevice));

    // Launch kernel with one block
    updateCMatrixKernel<<<1, number_of_Chebyshev_points>>>(d_K_stack, d_D_NN, d_C_NN);
    
    CUDA_CHECK(hipFree(d_D_NN));

    Eigen::VectorXd N_init(lambda_dimension/2);
    N_init << 1, 0, 0;

    Eigen::MatrixXd beta = Eigen::MatrixXd::Zero((lambda_dimension/2)*(number_of_Chebyshev_points-1),1);

    // Create Pointers
    double* d_Q_stack_CUDA = nullptr;
    double* d_Nbar_stack = nullptr;

    // Compute the memory occupation
    const auto size_of_Q_stack_CUDA_in_bytes = size_of_double * t_Q_stack_CUDA.size();
    const auto size_of_Nbar_stack_in_bytes = size_of_double * beta.size(); // Same dimension of beta (beta = -Nbar)

    // Allocate the memory
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_Q_stack_CUDA), size_of_Q_stack_CUDA_in_bytes));
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_Nbar_stack), size_of_Nbar_stack_in_bytes));

    //  Copy the data: hipMemcpy(destination, file_to_copy, size_of_the_file, std_cmd)
    CUDA_CHECK(hipMemcpy(d_Q_stack_CUDA, t_Q_stack_CUDA.data(), size_of_Q_stack_CUDA_in_bytes, hipMemcpyHostToDevice));

    // Launch the kernel: computeNbarKernel
    computeNbarKernel<<<1, number_of_Chebyshev_points - 1>>>(d_Q_stack_CUDA, d_Nbar_stack);

    //  Copy the data
    CUDA_CHECK(hipMemcpy(beta.data(), d_Nbar_stack, size_of_Nbar_stack_in_bytes, hipMemcpyDeviceToHost));

    CUDA_CHECK(hipFree(d_Q_stack_CUDA));
    CUDA_CHECK(hipFree(d_Nbar_stack));

    //Eigen::MatrixXd beta = -computeNbar(t_Q_stack_CUDA);

    //Definition of matrices dimensions.
    const int rows_C_NN = C_NN.rows();
    const int cols_C_NN = C_NN.cols();
    const int ld_C_NN = rows_C_NN;

    const int rows_D_IN = D_IN.rows();
    const int cols_D_IN = D_IN.cols();
    const int ld_D_IN = rows_D_IN;

    const int rows_N_init = N_init.rows();
    const int cols_N_init = N_init.cols();
    const int ld_N_init = rows_N_init;

    const int rows_beta = beta.rows();
    const int cols_beta = beta.cols();
    const int ld_beta = rows_beta;

    const int rows_N_stack = rows_beta;
    const int cols_N_stack = cols_beta;
    
    int info = 0;
    int lwork = 0;

    // Create Pointers
    // double* d_C_NN = nullptr;
    double* d_D_IN = nullptr;
    double* d_N_init = nullptr;
    double* d_beta = nullptr;
    double* d_N_stack = nullptr;
    double* d_work = nullptr;
    int* d_info = nullptr;

    // Compute the memory occupation
    const auto size_of_C_NN_in_bytes = size_of_double * C_NN.size();
    const auto size_of_D_IN_in_bytes = size_of_double * D_IN.size();
    const auto size_of_N_init_in_bytes = size_of_double * N_init.size();
    const auto size_of_beta_in_bytes = size_of_double * beta.size();
    const auto size_of_N_stack_in_bytes = size_of_double * rows_N_stack * cols_N_stack;

    // Allocate the memory
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_D_IN), size_of_D_IN_in_bytes));
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_N_init), size_of_N_init_in_bytes));
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_beta), size_of_beta_in_bytes));
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_N_stack), size_of_N_stack_in_bytes));
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_info), sizeof(int)));

    //  Copy the data: hipMemcpy(destination, file_to_copy, size_of_the_file, std_cmd)
    CUDA_CHECK(hipMemcpy(d_D_IN, D_IN.data(), size_of_D_IN_in_bytes, hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_N_init, N_init.data(), size_of_N_init_in_bytes, hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_beta, beta.data(), size_of_beta_in_bytes, hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_info, &info, sizeof(int), hipMemcpyHostToDevice));

    // Allocates buffer size for the LU decomposition
    hipsolverStatus_t status = hipsolverDnDgetrf_bufferSize(cusolverH, cols_C_NN, cols_C_NN, d_C_NN, ld_C_NN, &lwork);
    if (status != HIPSOLVER_STATUS_SUCCESS)
    {
        std::cerr << "cusolver error: " << getCusolverErrorString(status) << std::endl;
        // Handle or debug the error appropriately
    };

    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_work), size_of_double * lwork));

    //What we want to calculate
    Eigen::MatrixXd N_stack_CUDA(rows_N_stack, cols_N_stack);

    // res = -D_IN*N_init + beta
    double alpha_cublas = -1.0;
    double beta_cublas = 1.0;
    CUBLAS_CHECK(hipblasDgemm(cublasH, HIPBLAS_OP_N, HIPBLAS_OP_N, rows_D_IN, cols_N_init, cols_D_IN, &alpha_cublas, d_D_IN, ld_D_IN, d_N_init, ld_N_init, &beta_cublas, d_beta, ld_beta));

    // LU factorization
    CUSOLVER_CHECK(hipsolverDnDgetrf(cusolverH, cols_C_NN, cols_C_NN, d_C_NN, ld_C_NN, d_work, NULL, d_info));

    // Solving the final system
    CUSOLVER_CHECK(hipsolverDnDgetrs(cusolverH, HIPBLAS_OP_N, cols_C_NN, 1, d_C_NN, ld_C_NN, NULL, d_beta, ld_beta, d_info));

    CUDA_CHECK(hipMemcpy(N_stack_CUDA.data(), d_beta, size_of_beta_in_bytes, hipMemcpyDeviceToHost));

    //FREEING MEMORY
    CUDA_CHECK(
        hipFree(d_beta)
    );
    CUDA_CHECK(
        hipFree(d_C_NN)
    );
    CUDA_CHECK(
        hipFree(d_D_IN)
    );
    CUDA_CHECK(
        hipFree(d_info)
    );
    CUDA_CHECK(
        hipFree(d_N_init)
    );
    CUDA_CHECK(
        hipFree(d_N_stack)
    );
    CUDA_CHECK(
        hipFree(d_work)
    );


    return N_stack_CUDA;
}

__global__ void updateCouplesbKernel(const double* t_N_stack_CUDA, double* d_beta) {
    int idx = threadIdx.x;

    if (idx < number_of_Chebyshev_points - 1) {
        Eigen::VectorXd Gamma(lambda_dimension / 2);
        Gamma << 1, 0, 0;

        // Construct the skew-symmetric matrix manually
        Eigen::Matrix3d skewGamma;
        skewGamma << 0, -Gamma(2), Gamma(1),
                     Gamma(2), 0, -Gamma(0),
                    -Gamma(1), Gamma(0), 0;

        const Eigen::Vector3d C_bar = Eigen::Vector3d::Zero();
        Eigen::Vector3d N;

        int offset = idx * lambda_dimension / 2;

        for (int i = 0; i < lambda_dimension / 2; ++i) {
            N(i) = t_N_stack_CUDA[offset + i];
        }

        // Perform b = skewGamma.transpose() * N - C_bar
        double b[3] = { skewGamma[2]*N(1)-skewGamma[1]*N(2)-C_bar[0],
                            -skewGamma[2]*N(0)+skewGamma[0]*N(2)-C_bar[1],
                            skewGamma[1]*N(0)-skewGamma[0]*N(1)-C_bar[2]
                    };

        for (int i = 0; i < lambda_dimension / 2; ++i) {
            d_beta[offset + i] = b[i];
        }
    }
}

Eigen::MatrixXd integrateInternalCouples(Eigen::MatrixXd t_N_stack_CUDA)
{
    //  Now stack the matrices in the diagonal of bigger ones (as meny times as the state dimension)
    const Eigen::MatrixXd D_NN = Eigen::KroneckerProduct<Eigen::MatrixXd,Eigen::MatrixXd>(Eigen::MatrixXd::Identity(lambda_dimension/2, lambda_dimension/2), Dn_NN_B);
    const Eigen::MatrixXd D_IN = Eigen::KroneckerProduct<Eigen::MatrixXd,Eigen::MatrixXd>(Eigen::MatrixXd::Identity(lambda_dimension/2, lambda_dimension/2), Dn_IN_B);

    //Compute C_NN
    Eigen::MatrixXd C_NN = D_NN;
    
    // Compute the memory occupation 
    const auto size_of_D_NN_in_bytes = D_NN.size() * size_of_double;
    
    // Create Pointers
    double* d_D_NN = nullptr;
    double* d_C_NN = nullptr;

    // Allocate the memory
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_qe), size_of_qe_in_bytes));
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_D_NN), size_of_D_NN_in_bytes));
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_C_NN), size_of_D_NN_in_bytes));

    //  Copy the data
    CUDA_CHECK(hipMemcpy(d_D_NN, D_NN.data(), size_of_D_NN_in_bytes, hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_C_NN, C_NN.data(), size_of_D_NN_in_bytes, hipMemcpyHostToDevice));

    // Launch kernel with one block
    updateCMatrixKernel<<<1, number_of_Chebyshev_points-1>>>(d_K_stack, d_D_NN, d_C_NN);

    CUDA_CHECK(hipFree(d_D_NN));
    
    Eigen::MatrixXd beta_NN = Eigen::MatrixXd::Zero((lambda_dimension/2)*(number_of_Chebyshev_points-1), 1);

    // Create Pointers
    double* d_N_stack_CUDA = nullptr;
    double* d_beta_NN = nullptr;

    // Compute the memory occupation
    const auto size_of_N_stack_CUDA_in_bytes = size_of_double * t_N_stack_CUDA.size();
    const auto size_of_Nbar_stack_in_bytes = size_of_double * beta_NN.size(); // Same dimension of beta (beta = -Nbar)
    const auto size_of_beta_NN_in_bytes = size_of_double * beta_NN.size();


    // Allocate the memory
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_N_stack_CUDA), size_of_N_stack_CUDA_in_bytes));
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_beta_NN), size_of_beta_NN_in_bytes));

    //  Copy the data: hipMemcpy(destination, file_to_copy, size_of_the_file, std_cmd)
    CUDA_CHECK(hipMemcpy(d_N_stack_CUDA, t_N_stack_CUDA.data(), size_of_N_stack_CUDA_in_bytes, hipMemcpyHostToDevice));

    // Launch the kernel: computeNbarKernel
    updateCouplesbKernel<<<1, number_of_Chebyshev_points - 1>>>(d_N_stack_CUDA, d_beta_NN);

    // Free kernel memory
    CUDA_CHECK(hipFree(d_N_stack_CUDA));

    //  Copy the data
    CUDA_CHECK(hipMemcpy(beta_NN.data(), d_beta_NN, size_of_beta_NN_in_bytes, hipMemcpyDeviceToHost));

    Eigen::VectorXd C_init(lambda_dimension/2);
    C_init << 1, 0, 0;

    //What we want to calculate
    Eigen::MatrixXd C_stack_CUDA(t_N_stack_CUDA.rows(), t_N_stack_CUDA.cols());

    //Definition of matrices dimensions.
    const int rows_C_NN = C_NN.rows();
    const int cols_C_NN = C_NN.cols();
    const int ld_C_NN = rows_C_NN;

    const int rows_D_IN = D_IN.rows();
    const int cols_D_IN = D_IN.cols();
    const int ld_D_IN = rows_D_IN;

    const int rows_C_init = C_init.rows();
    const int cols_C_init = C_init.cols();
    const int ld_C_init = rows_C_init;

    const int rows_beta_NN = beta_NN.rows();
    const int cols_beta_NN = beta_NN.cols();
    const int ld_beta_NN = rows_beta_NN;
    
    int info = 0;
    int lwork = 0;

    // Create Pointers
    // double* d_C_NN = nullptr;
    double* d_D_IN = nullptr;
    double* d_C_init = nullptr;
    // double* d_beta_NN = nullptr;
    double* d_work = nullptr;
    int* d_info = nullptr;

    // Compute the memory occupation
    const auto size_of_C_NN_in_bytes = size_of_double * C_NN.size();
    const auto size_of_D_IN_in_bytes = size_of_double * D_IN.size();
    const auto size_of_C_init_in_bytes = size_of_double * C_init.size();
    // const auto size_of_beta_NN_in_bytes = size_of_double * beta_NN.size();
    const auto size_of_N_stack_in_bytes = size_of_double * t_N_stack_CUDA.size();

    // Allocate the memory
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_C_NN), size_of_C_NN_in_bytes));
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_D_IN), size_of_D_IN_in_bytes));
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_C_init), size_of_C_init_in_bytes));
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_beta_NN), size_of_beta_NN_in_bytes));
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_info), sizeof(int)));

    //  Copy the data: hipMemcpy(destination, file_to_copy, size_of_the_file, std_cmd)
    CUDA_CHECK(hipMemcpy(d_C_NN, C_NN.data(), size_of_C_NN_in_bytes, hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_D_IN, D_IN.data(), size_of_D_IN_in_bytes, hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_C_init, C_init.data(), size_of_C_init_in_bytes, hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_beta_NN, beta_NN.data(), size_of_beta_NN_in_bytes, hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_info, &info, sizeof(int), hipMemcpyHostToDevice));

    // Allocates buffer size for the LU decomposition
    hipsolverStatus_t status = hipsolverDnDgetrf_bufferSize(cusolverH, rows_C_NN, cols_C_NN, d_C_NN, ld_C_NN, &lwork);
    if (status != HIPSOLVER_STATUS_SUCCESS)
    {
        std::cerr << "cusolver error: " << getCusolverErrorString(status) << std::endl;
        // Handle or debug the error appropriately
    };

    //Has to be after hipsolverDnDgetrf_bufferSize as lwork is only computed then.
    CUDA_CHECK(
        hipMalloc(reinterpret_cast<void **>(&d_work), size_of_double * lwork)
    );

    double alpha_cublas = -1.0;
    double beta_cublas = 1.0;
    // res = -D_IN*C_init + beta_NN
    CUBLAS_CHECK(
        hipblasDgemm(cublasH, HIPBLAS_OP_N, HIPBLAS_OP_N, rows_D_IN, cols_C_init, cols_D_IN, &alpha_cublas, d_D_IN, ld_D_IN, d_C_init, ld_C_init, &beta_cublas, d_beta_NN, ld_beta_NN)
    );

    // LU factorization
    CUSOLVER_CHECK(
        hipsolverDnDgetrf(cusolverH, rows_C_NN, cols_C_NN, d_C_NN, ld_C_NN, d_work, NULL, d_info)
    );

    // Solving the final system
    CUSOLVER_CHECK(
        hipsolverDnDgetrs(cusolverH, HIPBLAS_OP_N, rows_C_NN, 1, d_C_NN, ld_C_NN, NULL, d_beta_NN, ld_beta_NN, d_info)
    );

    CUDA_CHECK(
        hipMemcpy(C_stack_CUDA.data(), d_beta_NN, size_of_beta_NN_in_bytes, hipMemcpyDeviceToHost)
    );

    //FREEING MEMORY
    CUDA_CHECK(
        hipFree(d_beta_NN)
    );
    CUDA_CHECK(
        hipFree(d_C_init)
    );
    CUDA_CHECK(
        hipFree(d_C_NN)
    );
    CUDA_CHECK(
        hipFree(d_D_IN)
    );
    CUDA_CHECK(
        hipFree(d_info)
    );
    CUDA_CHECK(
        hipFree(d_work)
    );

    return C_stack_CUDA;
}

Eigen::MatrixXd buildLambda(Eigen::MatrixXd t_C_stack_CUDA, Eigen::MatrixXd t_N_stack_CUDA)
{
    Eigen::Vector3d C;
    Eigen::Vector3d N;

    Eigen::VectorXd lambda(lambda_dimension);

    Eigen::MatrixXd Lambda_stack(lambda_dimension*(number_of_Chebyshev_points-1), 1);

    for (unsigned int i = 0; i < number_of_Chebyshev_points-1; ++i) {

        N << t_N_stack_CUDA(i),
             t_N_stack_CUDA(i  +  (number_of_Chebyshev_points-1)),
             t_N_stack_CUDA(i + 2*(number_of_Chebyshev_points-1));

        C << t_C_stack_CUDA(i),
             t_C_stack_CUDA(i  +  (number_of_Chebyshev_points-1)),
             t_C_stack_CUDA(i + 2*(number_of_Chebyshev_points-1));

        lambda << C, N;

        Lambda_stack.block<6,1>(i*lambda_dimension,0) = lambda;
    }

    return Lambda_stack;
}





// Used to build Qa_stack
// CUDA kernel function to update Qad_vector_b
__global__ void updateQad_vector_bKernel(double* t_Lambda_stack, double* B_NN) {
    int tid = threadIdx.x;

    // Define the Chebyshev points on the unit circle
    // ComputeChebyshevPoints function implementation here...


    if (tid < number_of_Chebyshev_points - 1) {
        // Create Eigen objects for B_NN and b
        Eigen::Map<Eigen::MatrixXd> B_NN_mat(B_NN, number_of_Chebyshev_points - 1, Qa_dimension);
        Eigen::VectorXd b(Qa_dimension);
        

        // Create Eigen object for B
        Eigen::Matrix<double, 6, 3> B;
        B.block(0, 0, 3, 3).setIdentity();
        B.block(3, 0, 3, 3).setZero();

        // Compute b for the current Chebyshev point
        for (int i = 0; i < lambda_dimension; ++i) {
            double* lambdaPtr = t_Lambda_stack + lambda_dimension * tid;
            Eigen::Map<Eigen::VectorXd> t_Lambda(lambdaPtr, lambda_dimension);

            b = -Phi_stack.block<na, na*ne>((tid+1)*(number_of_Chebyshev_points-1),0).transpose() * B.transpose() * t_Lambda;

            // Set the computed b in the B_NN matrix
            B_NN_mat.row(tid) = b.transpose();
        }
    }

}

Eigen::MatrixXd integrateGeneralisedForces(Eigen::MatrixXd t_Lambda_stack)
{
    // Qa_stack = B_NN*Dn_NN_inv
    Eigen::Vector3d Qa_init;
    Qa_init << 0,
               0,
               0;

    Eigen::MatrixXd B_NN(number_of_Chebyshev_points-1, Qa_dimension);

    // Dn_NN is constant so we can pre-invert
    Eigen::MatrixXd Dn_NN_inv = Dn_NN_B.inverse();

    // Compute the memory occupation 
    const auto size_of_B_NN_in_bytes = B_NN.size() * size_of_double;
    const auto size_of_Lambda_stack_in_bytes = t_Lambda_stack.size() * size_of_double;
    
    // Create Pointers
    double* d_B_NN = nullptr;
    double* d_Lambda_stack = nullptr;

    // Allocate the memory
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_B_NN), size_of_B_NN_in_bytes));
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_Lambda_stack), size_of_Lambda_stack_in_bytes));
    
    //  Copy the data
    CUDA_CHECK(hipMemcpy(d_Lambda_stack, t_Lambda_stack.data(), size_of_Lambda_stack_in_bytes, hipMemcpyHostToDevice));

    // Launch kernel with one block
    updateQad_vector_bKernel<<<1, number_of_Chebyshev_points-1>>>(d_Lambda_stack, d_B_NN);
 
    // Free kernel memory
    CUDA_CHECK(hipFree(t_Lambda_stack));

    //Definition of matrices dimensions.
    const int rows_B_NN = B_NN.rows();
    const int cols_B_NN = B_NN.cols();
    const int ld_B_NN = rows_B_NN;

    const int rows_Dn_NN_inv = Dn_NN_inv.rows();
    const int cols_Dn_NN_inv = Dn_NN_inv.cols();
    const int ld_Dn_NN_inv = rows_Dn_NN_inv;

    const int rows_Qa_stack = rows_Dn_NN_inv;
    const int cols_Qa_stack = cols_B_NN;
    const int ld_Qa_stack = rows_Qa_stack;

    // Create Pointers
    double* d_Dn_NN_inv = nullptr;    
    double* d_Qa_stack = nullptr;

    // Compute the memory occupation
    const auto size_of_Dn_NN_inv_in_bytes = size_of_double * Dn_NN_inv.size();
    const auto size_of_Qa_stack_in_bytes = size_of_double * rows_Qa_stack * cols_Qa_stack;

    // Allocate the memory
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_Dn_NN_inv), size_of_Dn_NN_inv_in_bytes));
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_Qa_stack), size_of_Qa_stack_in_bytes));

    //  Copy the data: hipMemcpy(destination, file_to_copy, size_of_the_file, std_cmd)
    CUDA_CHECK(hipMemcpy(d_Dn_NN_inv, Dn_NN_inv.data(), size_of_Dn_NN_inv_in_bytes, hipMemcpyHostToDevice));

    // Variable to check the result
    Eigen::MatrixXd Qa_stack_CUDA(rows_Qa_stack, cols_Qa_stack);

    // Compute Qa_stack = Dn_NN_inv*B_NN
    double alpha_cublas = 1.0;
    double beta_cublas = 0.0;
    CUBLAS_CHECK(
        hipblasDgemm(cublasH, HIPBLAS_OP_N, HIPBLAS_OP_N, rows_Dn_NN_inv, cols_B_NN, cols_Dn_NN_inv, &alpha_cublas, d_Dn_NN_inv, ld_Dn_NN_inv, d_B_NN, ld_B_NN, &beta_cublas, d_Qa_stack, ld_Qa_stack)
    );

    CUDA_CHECK(
        hipMemcpy(Qa_stack_CUDA.data(), d_Qa_stack, size_of_Qa_stack_in_bytes, hipMemcpyDeviceToHost));

    //FREEING MEMORY
    CUDA_CHECK(
        hipFree(d_B_NN)
    );
    CUDA_CHECK(
        hipFree(d_Qa_stack)
    );
    CUDA_CHECK(
        hipFree(d_Dn_NN_inv)
    );

    return Qa_stack_CUDA;
}






int main(int argc, char *argv[])
{

    //  cuda blas api need CUBLAS_CHECK
    CUBLAS_CHECK(
        hipblasCreate(&cublasH)
    );

    CUSOLVER_CHECK(
        hipsolverDnCreate(&cusolverH)
    );


    //  Here we give some value for the strain
    qe <<   0,
            0,
            0,
            1.2877691307032,
           -1.63807499160786,
            0.437406679142598,
            0,
            0,
            0;
    // qe.setZero();
    

    // Here we give some value for the strain rate
    for (unsigned int i = 0; i < number_of_Chebyshev_points; ++i) {
        Phi_stack.block<na, ne>(i*na, 0) =  Phi<na, ne>(Chebyshev_points[i]);
    }

    const auto Q_stack_CUDA = integrateQuaternions();
    std::cout << "Quaternion Integration : \n" << Q_stack_CUDA << std::endl;
    
    // const auto r_stack_CUDA = integratePosition(Q_stack_CUDA);
    // std::cout << "Position Integration : \n" << r_stack_CUDA << std::endl;

    // const auto N_stack_CUDA = integrateInternalForces(Q_stack_CUDA);
    // std::cout << "Internal Forces Integration : \n" << N_stack_CUDA << "\n" << std::endl;

    // const auto C_stack_CUDA = integrateInternalCouples(N_stack_CUDA);
    // std::cout << "Internal Couples Integration : \n" << C_stack_CUDA << "\n" << std::endl;

    // std::cout << "Internal Forces MATRIX : \n" << toMatrix(N_stack_CUDA, number_of_Chebyshev_points) << "\n" << std::endl;
    // std::cout << "Internal Couples MATRIX : \n" << toMatrix(C_stack_CUDA, number_of_Chebyshev_points) << "\n" << std::endl;
    
    // const auto Lambda_stack_CUDA = buildLambda(C_stack_CUDA, N_stack_CUDA);
    // //std::cout << "Lambda_stack : \n" << Lambda_stack_CUDA << "\n" << std::endl;

    // const auto Qa_stack_CUDA = integrateGeneralisedForces(Lambda_stack_CUDA);
    // std::cout << "Generalized Forces Integration : \n" << Qa_stack_CUDA << std::endl;

    /*
    Destry cuda objects
    */
    CUBLAS_CHECK(
        hipblasDestroy(cublasH)
    );

    CUSOLVER_CHECK(
        hipsolverDnDestroy(cusolverH)
    );

    CUDA_CHECK(
        hipDeviceReset()
    );

    return 0;
}
