#include <iostream>
#include <benchmark/benchmark.h>
#include <cstdio>
#include <cstdlib>
#include <vector>
#include <chrono>

//  CUDA Basic Linear Algebra 
#include <hipblas.h>

#include "utilities.h"  //  Utilities for error handling (must be after cublas or cusolver)
#include "getCusolverErrorString.h"
#include "benchmark_csv_exporter.h"


#include <Eigen/Dense>
#include <Eigen/LU>



void benchmarkMatMul_CPU(::benchmark::State &t_state)
{
    const unsigned int dim = t_state.range(0);
    Eigen::MatrixXd A = Eigen::MatrixXd::Random(dim, dim);

    t_state.counters = {
      {"dim: ", dim},
    };


    while(t_state.KeepRunning()){
        Eigen::PartialPivLU<Eigen::MatrixXd> lu(A); // Perform LU factorization
    }
    //exportBenchmarkResultsToCSV(benchmark1_name + ".csv", .name(), .iterations(), t_state.real_time(), t_state.cpu_time());
};


void benchmarkMatMul_GPU(::benchmark::State &t_state)
{
    int dim = t_state.range(0);

    t_state.counters = {
      {"dim: ", dim},
    };

    hipsolverHandle_t cusolverH = NULL;
    hipblasHandle_t cublasH = NULL;

    CUBLAS_CHECK(
        hipblasCreate(&cublasH)
    );

    CUSOLVER_CHECK(
        hipsolverDnCreate(&cusolverH)
    );

    Eigen::MatrixXd A = Eigen::MatrixXd::Random(dim, dim);

    const auto size_of_double = sizeof(double);

    // LU factorization variables
    int info = 0;
    int lwork = 0;

    const int rows_A = A.rows();
    const int cols_A = A.cols();
    const int ld_A = rows_A;

    double*  d_A = nullptr;
    double* d_work = nullptr;
    int* d_info = nullptr;

    // Compute the memory occupation (I commented out the memory occupation for res in the following.)
    const auto size_of_A_in_bytes = size_of_double * A.size();

    // Allocate the memory
    CUDA_CHECK(
        hipMalloc(reinterpret_cast<void **>(&d_A), size_of_A_in_bytes)
    );
    CUDA_CHECK(
        hipMalloc(reinterpret_cast<void **>(&d_info), sizeof(int))
    );
    
    //  Copy the data: hipMemcpy(destination, file_to_copy, size_of_the_file, std_cmd)
    CUDA_CHECK(
        hipMemcpy(d_A, A.data(), size_of_A_in_bytes, hipMemcpyHostToDevice)
    );
    CUDA_CHECK(
        hipMemcpy(d_info, &info, sizeof(int), hipMemcpyHostToDevice)
    );

    // Allocates buffer size for the LU decomposition
    hipsolverStatus_t status = hipsolverDnDgetrf_bufferSize(cusolverH, rows_A, cols_A, d_A, ld_A, &lwork);
    if (status != HIPSOLVER_STATUS_SUCCESS)
    {
        std::cerr << "cusolver error: " << getCusolverErrorString(status) << std::endl;
    };
    // Allocate the memory for LU factorization workspace
    CUDA_CHECK(
        hipMalloc(reinterpret_cast<void **>(&d_work), size_of_double * lwork)
    );

    // //Template
    // for (auto _ : t_state) {
    //     auto start = std::chrono::high_resolution_clock::now();
    //     //insert benchmark code here
    //     auto end = std::chrono::high_resolution_clock::now();

    //     auto elapsed_seconds = std::chrono::duration_cast<std::chrono::duration<double>>(end - start);

    //     t_state.SetIterationTime(elapsed_seconds.count());
    // }

    for (auto _ : t_state) {
        auto start = std::chrono::high_resolution_clock::now();
        // LU factorization
        CUSOLVER_CHECK(
            hipsolverDnDgetrf(cusolverH, rows_A, cols_A, d_A, ld_A, d_work, NULL, d_info)
        );
        CUDA_CHECK(
            hipMemcpy(A.data(), d_A, size_of_A_in_bytes, hipMemcpyDeviceToHost)
        );
        auto end = std::chrono::high_resolution_clock::now();

        auto elapsed_seconds = std::chrono::duration_cast<std::chrono::duration<double>>(end - start);

        t_state.SetIterationTime(elapsed_seconds.count());
    }

    //exportBenchmarkResultsToCSV(benchmark2_name + ".csv", t_state.name(), t_state.iterations(), t_state.real_time(), t_state.cpu_time());
   
    CUDA_CHECK(
        hipFree(d_A)
    );
    CUDA_CHECK(
        hipFree(d_info)
    );
        CUDA_CHECK(
        hipFree(d_work)
    );
    CUBLAS_CHECK(
        hipblasDestroy(cublasH)
    );
    CUSOLVER_CHECK(
        hipsolverDnDestroy(cusolverH)
    );
    CUDA_CHECK(
        hipDeviceReset()
    );
};



int main(int argc, char *argv[])
{

    const unsigned int repetitions = 20;

    std::vector<unsigned int> matrix_dim = {20, 50, 100, 150, 200, 250, 300, 350, 400, 450, 500};

    const std::string benchmark1_name = "LU factorization CPU";
    const std::string benchmark2_name = "LU factorization GPU";


    for(const auto dim : matrix_dim)
        ::benchmark::RegisterBenchmark(benchmark1_name.c_str(), benchmarkMatMul_CPU)->Arg(dim)->Repetitions(repetitions)->Unit(::benchmark::kMicrosecond);
        
    ::benchmark::RegisterBenchmark(benchmark1_name.c_str(), [](::benchmark::State &t_state){
        for(auto _ : t_state)
            int a = 0;
    });

    for(const auto dim : matrix_dim)
        ::benchmark::RegisterBenchmark(benchmark2_name.c_str(), benchmarkMatMul_GPU)->Arg(dim)->Repetitions(repetitions)->Unit(::benchmark::kMicrosecond);

    ::benchmark::RegisterBenchmark(benchmark2_name.c_str(), [](::benchmark::State &t_state){
        for(auto _ : t_state)
            int a = 0;
    });
            



    ::benchmark::Initialize(&argc, argv);


    ::benchmark::RunSpecifiedBenchmarks();


    return 0;
}
