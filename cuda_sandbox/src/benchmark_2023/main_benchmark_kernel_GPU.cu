#include "hip/hip_runtime.h"
#include <cstdio>
#include <cstdlib>
#include <vector>

#include <fstream>
#include <cmath>

#include <hipblas.h>
// // #include <hip/hip_runtime.h>

#include "spectral_integration_utilities.h"
#include "chebyshev_differentiation.h"
#include "lie_algebra_utilities.h"
#include "utilities.h"  //  Utilities for error handling (must be after cublas or cusolver)
#include "globals.h"
#include "getCusolverErrorString.h"

#include <Eigen/Dense>
#include <unsupported/Eigen/KroneckerProduct>
#include <benchmark/benchmark.h>


//GLOBAL VARIABLES ARE DEFINED IN globals.h

__device__ void quaternionToRotationMatrix(const double* q, double* R) {
    double q0 = q[0];
    double q1 = q[1];
    double q2 = q[2];
    double q3 = q[3];

    R[0] = q0 * q0 + q1 * q1 - q2 * q2 - q3 * q3;
    R[1] = 2.0 * (q1 * q2 - q0 * q3);
    R[2] = 2.0 * (q1 * q3 + q0 * q2);
    R[3] = 2.0 * (q1 * q2 + q0 * q3);
    R[4] = q0 * q0 - q1 * q1 + q2 * q2 - q3 * q3;
    R[5] = 2.0 * (q2 * q3 - q0 * q1);
    R[6] = 2.0 * (q1 * q3 - q0 * q2);
    R[7] = 2.0 * (q2 * q3 + q0 * q1);
    R[8] = q0 * q0 - q1 * q1 - q2 * q2 + q3 * q3;
}


// Used to build Q_stack
__global__ void computeCMatrixKernel(const double* d_K_stack, const double* D_NN, double* C_NN) {

    int i = threadIdx.x;

    #pragma region Compute_C_NN
    if (i < number_of_Chebyshev_points-1) {
        int row = 0;
        int col = 1;
        int row_index = row * (number_of_Chebyshev_points - 1) + i;
        int col_index = col * (number_of_Chebyshev_points - 1) + i;
        C_NN[row_index * quaternion_state_dimension + col_index] = D_NN[row_index * quaternion_state_dimension + col_index] + 0.5*d_K_stack[3*i+0];


        row = 0;
        col = 2;
        row_index = row * (number_of_Chebyshev_points - 1) + i;
        col_index = col * (number_of_Chebyshev_points - 1) + i;
        C_NN[row_index * quaternion_state_dimension + col_index] = D_NN[row_index * quaternion_state_dimension + col_index] + 0.5*d_K_stack[3*i+1];

        row = 0;
        col = 3;
        row_index = row * (number_of_Chebyshev_points - 1) + i;
        col_index = col * (number_of_Chebyshev_points - 1) + i;
        C_NN[row_index * quaternion_state_dimension + col_index] = D_NN[row_index * quaternion_state_dimension + col_index] + 0.5*d_K_stack[3*i+2];

        row = 1;
        col = 0;
        row_index = row * (number_of_Chebyshev_points - 1) + i;
        col_index = col * (number_of_Chebyshev_points - 1) + i;
        C_NN[row_index * quaternion_state_dimension + col_index] = D_NN[row_index * quaternion_state_dimension + col_index] - 0.5*d_K_stack[3*i+0];

        row = 1;
        col = 2;
        row_index = row * (number_of_Chebyshev_points - 1) + i;
        col_index = col * (number_of_Chebyshev_points - 1) + i;
        C_NN[row_index * quaternion_state_dimension + col_index] = D_NN[row_index * quaternion_state_dimension + col_index] - 0.5*d_K_stack[3*i+2];

        row = 1;
        col = 3;
        row_index = row * (number_of_Chebyshev_points - 1) + i;
        col_index = col * (number_of_Chebyshev_points - 1) + i;
        C_NN[row_index * quaternion_state_dimension + col_index] = D_NN[row_index * quaternion_state_dimension + col_index] + 0.5*d_K_stack[3*i+1];

        row = 2;
        col = 0;
        row_index = row * (number_of_Chebyshev_points - 1) + i;
        col_index = col * (number_of_Chebyshev_points - 1) + i;
        C_NN[row_index * quaternion_state_dimension + col_index] = D_NN[row_index * quaternion_state_dimension + col_index] - 0.5*d_K_stack[3*i+1];

        row = 2;
        col = 1;
        row_index = row * (number_of_Chebyshev_points - 1) + i;
        col_index = col * (number_of_Chebyshev_points - 1) + i;
        C_NN[row_index * quaternion_state_dimension + col_index] = D_NN[row_index * quaternion_state_dimension + col_index] + 0.5*d_K_stack[3*i+2];

        row = 2;
        col = 3;
        row_index = row * (number_of_Chebyshev_points - 1) + i;
        col_index = col * (number_of_Chebyshev_points - 1) + i;
        C_NN[row_index * quaternion_state_dimension + col_index] = D_NN[row_index * quaternion_state_dimension + col_index] - 0.5*d_K_stack[3*i+0];

        row = 3;
        col = 0;
        row_index = row * (number_of_Chebyshev_points - 1) + i;
        col_index = col * (number_of_Chebyshev_points - 1) + i;
        C_NN[row_index * quaternion_state_dimension + col_index] = D_NN[row_index * quaternion_state_dimension + col_index] - 0.5*d_K_stack[3*i+2];

        row = 3;
        col = 1;
        row_index = row * (number_of_Chebyshev_points - 1) + i;
        col_index = col * (number_of_Chebyshev_points - 1) + i;
        C_NN[row_index * quaternion_state_dimension + col_index] = D_NN[row_index * quaternion_state_dimension + col_index] - 0.5*d_K_stack[3*i+1];

        row = 3;
        col = 2;
        row_index = row * (number_of_Chebyshev_points - 1) + i;
        col_index = col * (number_of_Chebyshev_points - 1) + i;
        C_NN[row_index * quaternion_state_dimension + col_index] = D_NN[row_index * quaternion_state_dimension + col_index] + 0.5*d_K_stack[3*i+0];
    }
    #pragma endregion

}

Eigen::VectorXd integrateQuaternions()
{       
    ::benchmark::RegisterBenchmark("Integrate Quaternions:", [&](::benchmark::State &t_state){

    t_state.counters = {
        {"na", na},
        {"ne", ne},
        {"Cheb pts", number_of_Chebyshev_points}
    };

    // INITIALISATION
    #pragma region K_stack

    // Allocate memory on the device
    CUDA_CHECK(hipMalloc((void**)&d_Phi_stack, size_of_Phi_stack_in_bytes));
    CUDA_CHECK(hipMalloc((void**)&d_qe, size_of_qe_in_bytes));
    CUDA_CHECK(hipMalloc((void**)&d_K_stack, size_of_K_stack_in_bytes));

    // Copy input data from host to device
    CUDA_CHECK(hipMemcpy(d_Phi_stack, Phi_stack.data(), size_of_Phi_stack_in_bytes, hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_qe, qe.data(), size_of_qe_in_bytes, hipMemcpyHostToDevice));

    // Define stride values
    int ld_Phi_stack = na; // Leading dimension of matrix A
    int inc_qe = 1; // Increment for x
    int inc_K_stack = 1; // Increment for y
    int stride_Phi_stack = na * (na * ne); // Stride between matrices in A
    int stride_qe = 0; // Stride between vectors in x
    int stride_K_stack = na; // Stride between vectors in y

    // Perform gemvStridedBatched operation
    double alpha_cublas = 1.0; // Scalar alpha
    double beta_cublas = 0.0; // Scalar beta
    CUBLAS_CHECK(hipblasDgemvStridedBatched(cublasH, HIPBLAS_OP_N, na, na*ne, &alpha_cublas, d_Phi_stack, ld_Phi_stack, stride_Phi_stack,
                                            d_qe, inc_qe, stride_qe, &beta_cublas, d_K_stack, inc_K_stack, stride_K_stack, number_of_Chebyshev_points));
    
    #pragma endregion

    // Vectors definitions
    const Eigen::MatrixXd D_NN = Eigen::KroneckerProduct<Eigen::MatrixXd,Eigen::MatrixXd>(Eigen::MatrixXd::Identity(quaternion_state_dimension, quaternion_state_dimension), Dn_NN_F);
    const Eigen::MatrixXd D_IN = Eigen::KroneckerProduct<Eigen::MatrixXd,Eigen::MatrixXd>(Eigen::MatrixXd::Identity(quaternion_state_dimension, quaternion_state_dimension), Dn_IN_F);

    Eigen::MatrixXd C_NN = D_NN;

    Eigen::MatrixXd q_init(4,1);
    q_init << 1, 0, 0, 0;

    Eigen::MatrixXd b = Eigen::MatrixXd::Zero(quaternion_problem_dimension,1);

    // Dimension definition
    const int rows_D_IN = D_IN.rows();
    const int cols_D_IN = D_IN.cols();
    const int ld_D_IN = rows_D_IN;  

    const int rows_q_init = q_init.rows();
    const int cols_q_init = q_init.cols();
    const int ld_q_init = rows_q_init;

    const int rows_b = b.rows();
    const int cols_b = b.cols();
    const int ld_b = rows_b;

    const int rows_res = b.rows();
    const int cols_res = b.cols();
    
    const int rows_C_NN = C_NN.rows();
    const int cols_C_NN = C_NN.cols();
    const int ld_C_NN = rows_C_NN;

    const int rows_Q_stack = rows_C_NN;
    const int cols_Q_stack = cols_b;

    Eigen::MatrixXd Q_stack_CUDA(rows_Q_stack, cols_Q_stack); //What we want to calculate

    // LU factorization variables
    int info = 0;
    int lwork = 0;
    
    // Compute the memory occupation 
    const auto size_of_D_IN_in_bytes = size_of_double * D_IN.size();
    const auto size_of_q_init_in_bytes = size_of_double * q_init.size();
    const auto size_of_b_in_bytes = size_of_double * b.size();
    const auto size_of_res_in_bytes = size_of_double * rows_res * cols_res;
    const auto size_of_Q_stack_in_bytes = size_of_double * rows_Q_stack * cols_Q_stack;
    const auto size_of_D_NN_in_bytes = D_NN.size() * size_of_double;
    
    // Create Pointers for computeCMatrixKernel
    double* d_D_NN = nullptr;
    double* d_C_NN = nullptr;
    // Create Pointers
    double* d_D_IN = nullptr;
    double* d_q_init = nullptr;
    double* d_b = nullptr;
    double* d_res = nullptr;
    double* d_Q_stack = nullptr;
    double* d_work = nullptr;
    int* d_info = nullptr;


    // Allocate the memory for computeCMatrixKernel
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_qe), size_of_qe_in_bytes));
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_D_NN), size_of_D_NN_in_bytes));
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_C_NN), size_of_D_NN_in_bytes)); // same size of D_NN
    // Allocate the memory
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_D_IN), size_of_D_IN_in_bytes));
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_q_init), size_of_q_init_in_bytes));
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_b), size_of_b_in_bytes));
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_res), size_of_res_in_bytes));
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_Q_stack), size_of_Q_stack_in_bytes));
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_info), sizeof(int)));

    // Allocates buffer size for the LU decomposition
    hipsolverStatus_t status = hipsolverDnDgetrf_bufferSize(cusolverH, rows_C_NN, cols_Q_stack, d_C_NN, ld_C_NN, &lwork);
    if (status != HIPSOLVER_STATUS_SUCCESS)
    {
        std::cerr << "cusolver error: " << getCusolverErrorString(status) << std::endl;
    };
    // Allocate the memory for LU factorization workspace
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_work), size_of_double * lwork));

    //  Copy the data for computeCMatrixKernel
    CUDA_CHECK(hipMemcpy(d_D_NN, D_NN.data(), size_of_D_NN_in_bytes, hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_C_NN, C_NN.data(), size_of_D_NN_in_bytes, hipMemcpyHostToDevice));
    //  Copy the data
    CUDA_CHECK(hipMemcpy(d_D_IN, D_IN.data(), size_of_D_IN_in_bytes, hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_q_init, q_init.data(), size_of_q_init_in_bytes, hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_b, b.data(), size_of_b_in_bytes, hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_info, &info, sizeof(int), hipMemcpyHostToDevice));

    //  BENCHMARK
    for (auto _ : t_state) {
        auto start = std::chrono::high_resolution_clock::now();
        // TO BENCHMARK: START
        // Launch kernel with one block
        computeCMatrixKernel<<<1, number_of_Chebyshev_points-1>>>(d_K_stack, d_D_NN, d_C_NN);

        // Computing b = -D_IN*q_init + b
        alpha_cublas = -1.0;
        beta_cublas = 1.0;
        CUBLAS_CHECK(hipblasDgemm(cublasH, HIPBLAS_OP_N, HIPBLAS_OP_N, rows_D_IN, cols_q_init, cols_D_IN, &alpha_cublas, d_D_IN, ld_D_IN, d_q_init, ld_q_init, &beta_cublas, d_b, ld_b));

        // LU factorization
        CUSOLVER_CHECK(hipsolverDnDgetrf(cusolverH, rows_C_NN, cols_C_NN, d_C_NN, ld_C_NN, d_work, NULL, d_info));

        // Solving the final system
        CUSOLVER_CHECK(hipsolverDnDgetrs(cusolverH, HIPBLAS_OP_N, rows_C_NN, 1, d_C_NN, ld_C_NN, NULL, d_b, ld_b, d_info));
        // TO BENCHMARK: END
        auto end = std::chrono::high_resolution_clock::now();
        auto elapsed_seconds = std::chrono::duration_cast<std::chrono::duration<double>>(end - start);

        t_state.SetIterationTime(elapsed_seconds.count());
    }

    // Memory Copy
    CUDA_CHECK(hipMemcpy(Q_stack_CUDA.data(), d_b, size_of_b_in_bytes, hipMemcpyDeviceToHost));

    // FREEING MEMORY
    CUDA_CHECK(hipFree(d_D_NN));
    CUDA_CHECK(hipFree(d_D_IN));
    CUDA_CHECK(hipFree(d_C_NN));
    CUDA_CHECK(hipFree(d_b));
    CUDA_CHECK(hipFree(d_info));
    CUDA_CHECK(hipFree(d_q_init));
    CUDA_CHECK(hipFree(d_Q_stack));
    CUDA_CHECK(hipFree(d_res));
    CUDA_CHECK(hipFree(d_work));

    })->Repetitions(20)->Unit(::benchmark::kMicrosecond);

// INITIALISATION
    #pragma region integrateQuaternions
    #pragma region K_stack

    // Allocate memory on the device
    CUDA_CHECK(hipMalloc((void**)&d_Phi_stack, size_of_Phi_stack_in_bytes));
    CUDA_CHECK(hipMalloc((void**)&d_qe, size_of_qe_in_bytes));
    CUDA_CHECK(hipMalloc((void**)&d_K_stack, size_of_K_stack_in_bytes));

    // Copy input data from host to device
    CUDA_CHECK(hipMemcpy(d_Phi_stack, Phi_stack.data(), size_of_Phi_stack_in_bytes, hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_qe, qe.data(), size_of_qe_in_bytes, hipMemcpyHostToDevice));

    // Define stride values
    int ld_Phi_stack = na; // Leading dimension of matrix A
    int inc_qe = 1; // Increment for x
    int inc_K_stack = 1; // Increment for y
    int stride_Phi_stack = na * (na * ne); // Stride between matrices in A
    int stride_qe = 0; // Stride between vectors in x
    int stride_K_stack = na; // Stride between vectors in y

    // Perform gemvStridedBatched operation
    double alpha_cublas = 1.0; // Scalar alpha
    double beta_cublas = 0.0; // Scalar beta
    CUBLAS_CHECK(hipblasDgemvStridedBatched(cublasH, HIPBLAS_OP_N, na, na*ne, &alpha_cublas, d_Phi_stack, ld_Phi_stack, stride_Phi_stack,
                                            d_qe, inc_qe, stride_qe, &beta_cublas, d_K_stack, inc_K_stack, stride_K_stack, number_of_Chebyshev_points));
    
    #pragma endregion

    // Vectors definitions
    const Eigen::MatrixXd D_NN = Eigen::KroneckerProduct<Eigen::MatrixXd,Eigen::MatrixXd>(Eigen::MatrixXd::Identity(quaternion_state_dimension, quaternion_state_dimension), Dn_NN_F);
    const Eigen::MatrixXd D_IN = Eigen::KroneckerProduct<Eigen::MatrixXd,Eigen::MatrixXd>(Eigen::MatrixXd::Identity(quaternion_state_dimension, quaternion_state_dimension), Dn_IN_F);

    Eigen::MatrixXd C_NN = D_NN;

    Eigen::MatrixXd q_init(4,1);
    q_init << 1, 0, 0, 0;

    Eigen::MatrixXd b = Eigen::MatrixXd::Zero(quaternion_problem_dimension,1);

    // Dimension definition
    const int rows_D_IN = D_IN.rows();
    const int cols_D_IN = D_IN.cols();
    const int ld_D_IN = rows_D_IN;  

    const int rows_q_init = q_init.rows();
    const int cols_q_init = q_init.cols();
    const int ld_q_init = rows_q_init;

    const int rows_b = b.rows();
    const int cols_b = b.cols();
    const int ld_b = rows_b;

    const int rows_res = b.rows();
    const int cols_res = b.cols();
    
    const int rows_C_NN = C_NN.rows();
    const int cols_C_NN = C_NN.cols();
    const int ld_C_NN = rows_C_NN;

    const int rows_Q_stack = rows_C_NN;
    const int cols_Q_stack = cols_b;

    Eigen::MatrixXd Q_stack_CUDA(rows_Q_stack, cols_Q_stack); //What we want to calculate

    // LU factorization variables
    int info = 0;
    int lwork = 0;
    
    // Compute the memory occupation 
    const auto size_of_D_IN_in_bytes = size_of_double * D_IN.size();
    const auto size_of_q_init_in_bytes = size_of_double * q_init.size();
    const auto size_of_b_in_bytes = size_of_double * b.size();
    const auto size_of_res_in_bytes = size_of_double * rows_res * cols_res;
    const auto size_of_Q_stack_in_bytes = size_of_double * rows_Q_stack * cols_Q_stack;
    const auto size_of_D_NN_in_bytes = D_NN.size() * size_of_double;
    
    // Create Pointers for computeCMatrixKernel
    double* d_D_NN = nullptr;
    double* d_C_NN = nullptr;
    // Create Pointers
    double* d_D_IN = nullptr;
    double* d_q_init = nullptr;
    double* d_b = nullptr;
    double* d_res = nullptr;
    double* d_Q_stack = nullptr;
    double* d_work = nullptr;
    int* d_info = nullptr;


    // Allocate the memory for computeCMatrixKernel
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_qe), size_of_qe_in_bytes));
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_D_NN), size_of_D_NN_in_bytes));
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_C_NN), size_of_D_NN_in_bytes)); // same size of D_NN
    // Allocate the memory
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_D_IN), size_of_D_IN_in_bytes));
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_q_init), size_of_q_init_in_bytes));
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_b), size_of_b_in_bytes));
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_res), size_of_res_in_bytes));
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_Q_stack), size_of_Q_stack_in_bytes));
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_info), sizeof(int)));

    // Allocates buffer size for the LU decomposition
    hipsolverStatus_t status = hipsolverDnDgetrf_bufferSize(cusolverH, rows_C_NN, cols_Q_stack, d_C_NN, ld_C_NN, &lwork);
    if (status != HIPSOLVER_STATUS_SUCCESS)
    {
        std::cerr << "cusolver error: " << getCusolverErrorString(status) << std::endl;
    };
    // Allocate the memory for LU factorization workspace
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_work), size_of_double * lwork));

    //  Copy the data for computeCMatrixKernel
    CUDA_CHECK(hipMemcpy(d_D_NN, D_NN.data(), size_of_D_NN_in_bytes, hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_C_NN, C_NN.data(), size_of_D_NN_in_bytes, hipMemcpyHostToDevice));
    //  Copy the data
    CUDA_CHECK(hipMemcpy(d_D_IN, D_IN.data(), size_of_D_IN_in_bytes, hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_q_init, q_init.data(), size_of_q_init_in_bytes, hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_b, b.data(), size_of_b_in_bytes, hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_info, &info, sizeof(int), hipMemcpyHostToDevice));

    // Launch kernel with one block
    computeCMatrixKernel<<<1, number_of_Chebyshev_points-1>>>(d_K_stack, d_D_NN, d_C_NN);

    // Computing b = -D_IN*q_init + b
    alpha_cublas = -1.0;
    beta_cublas = 1.0;
    CUBLAS_CHECK(hipblasDgemm(cublasH, HIPBLAS_OP_N, HIPBLAS_OP_N, rows_D_IN, cols_q_init, cols_D_IN, &alpha_cublas, d_D_IN, ld_D_IN, d_q_init, ld_q_init, &beta_cublas, d_b, ld_b));

    // LU factorization
    CUSOLVER_CHECK(hipsolverDnDgetrf(cusolverH, rows_C_NN, cols_C_NN, d_C_NN, ld_C_NN, d_work, NULL, d_info));

    // Solving the final system
    CUSOLVER_CHECK(hipsolverDnDgetrs(cusolverH, HIPBLAS_OP_N, rows_C_NN, 1, d_C_NN, ld_C_NN, NULL, d_b, ld_b, d_info));

    // Memory Copy
    CUDA_CHECK(hipMemcpy(Q_stack_CUDA.data(), d_b, size_of_b_in_bytes, hipMemcpyDeviceToHost));

    // FREEING MEMORY
    CUDA_CHECK(hipFree(d_D_NN));
    CUDA_CHECK(hipFree(d_D_IN));
    CUDA_CHECK(hipFree(d_C_NN));
    CUDA_CHECK(hipFree(d_b));
    CUDA_CHECK(hipFree(d_info));
    CUDA_CHECK(hipFree(d_q_init));
    CUDA_CHECK(hipFree(d_Q_stack));
    CUDA_CHECK(hipFree(d_res));
    CUDA_CHECK(hipFree(d_work));

    #pragma endregion integrateQuaternions

    return Q_stack_CUDA;
}



// Used to build r_stack
__global__ void computeIvpKernel(double* t_Dn_IN_F, double* t_r_init, double* t_ivp) {
    int i = threadIdx.x;

    if (i < number_of_Chebyshev_points-1) {
        t_ivp[i] = t_Dn_IN_F[i] * t_r_init[0];
        t_ivp[i+1] = t_Dn_IN_F[i] * t_r_init[1];
        t_ivp[i+2] = t_Dn_IN_F[i] * t_r_init[2];
    }
}

__global__ void updatePositionbKernel(double* d_Q_stack_CUDA, double* d_b, double* d_ivp){
    
    int i = threadIdx.x;

    if (i < number_of_Chebyshev_points-1) {
        double q[4] = { d_Q_stack_CUDA[i], 
                        d_Q_stack_CUDA[i + (number_of_Chebyshev_points-1)],
                        d_Q_stack_CUDA[i + 2*(number_of_Chebyshev_points-1)], 
                        d_Q_stack_CUDA[i + 3*(number_of_Chebyshev_points-1)]
                        };

        double R[9];

        quaternionToRotationMatrix(q, R);

        // b.block<1, 3>(i, 0) = (Eigen::Map<Eigen::MatrixXd>(R, 3, 3) * Eigen::Vector3d(1, 0, 0)).transpose();
        d_b[0+i*position_dimension] = R[0] - d_ivp[0+i*position_dimension];
        d_b[1+i*position_dimension] = R[3] - d_ivp[1+i*position_dimension];
        d_b[2+i*position_dimension] = R[6] - d_ivp[2+i*position_dimension];

    }
}

Eigen::MatrixXd integratePosition(Eigen::MatrixXd t_Q_stack_CUDA)
{       
    ::benchmark::RegisterBenchmark("Integrate Position:", [&](::benchmark::State &t_state){

    t_state.counters = {
        {"na", na},
        {"ne", ne},
        {"Cheb pts", number_of_Chebyshev_points}
    };

    // INITIALISATION
    // Vectors definitions
    Eigen::Vector3d r_init;
    r_init << 0,
              0,
              0;

    Eigen::MatrixXd Dn_NN_inv = Dn_NN_F.inverse(); //  This matrix remains constant so we can pre invert

    Eigen::Matrix<double, number_of_Chebyshev_points-1, position_dimension> b; //used in updatePositionbKernel
    
    Eigen::MatrixXd ivp(number_of_Chebyshev_points-1, position_dimension);

    Eigen::MatrixXd res(number_of_Chebyshev_points-1, position_dimension);

    // Dimension definition
    const int rows_Dn_NN_inv = Dn_NN_inv.rows();
    const int cols_Dn_NN_inv = Dn_NN_inv.cols();
    const int ld_Dn_NN_inv = rows_Dn_NN_inv;  

    const int rows_res = res.rows();
    const int cols_res = res.cols();
    const int ld_res = rows_res;

    const int rows_r_stack = rows_Dn_NN_inv;
    const int cols_r_stack = cols_res;
    const int ld_r_stack = rows_r_stack;

    Eigen::MatrixXd r_stack_CUDA(rows_r_stack, cols_r_stack); // What we want to calculate 

    // Compute the memory occupation for computeIvpKernel/updatePositionbKernel
    const auto size_of_Q_stack_CUDA_in_bytes = t_Q_stack_CUDA.size() * size_of_double;
    const auto size_of_b_in_bytes = b.size() * size_of_double;
    const auto size_of_r_init_in_bytes = r_init.size() * size_of_double;
    const auto size_of_ivp_in_bytes = ivp.size() * size_of_double;
    const auto size_of_Dn_IN_F_in_bytes = Dn_IN_F.size() * size_of_double;
    // Compute the memory occupation
    const auto size_of_Dn_NN_inv_in_bytes = size_of_double * Dn_NN_inv.size();
    const auto size_of_r_stack_in_bytes = size_of_double * rows_r_stack * cols_r_stack;

    // Create Pointers for computeIvpKernel/updatePositionbKernel
    double* d_Q_stack_CUDA;
    double* d_b;
    double* d_r_init;
    double* d_ivp;
    double* d_Dn_IN_F;
    // Create Pointers
    double* d_Dn_NN_inv = nullptr;
    double* d_r_stack = nullptr;

    // Allocate the memory for computeIvpKernel/updatePositionbKernel
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_Q_stack_CUDA), size_of_Q_stack_CUDA_in_bytes));
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_b), size_of_b_in_bytes));
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_r_init), size_of_r_init_in_bytes));
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_ivp), size_of_ivp_in_bytes));
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_Dn_IN_F), size_of_Dn_IN_F_in_bytes));
    // Allocate the memory
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_Dn_NN_inv), size_of_Dn_NN_inv_in_bytes));
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_r_stack), size_of_r_stack_in_bytes));

    //  Copy the data for computeIvpKernel/updatePositionbKernel
    CUDA_CHECK(hipMemcpy(d_Q_stack_CUDA, t_Q_stack_CUDA.data(), size_of_Q_stack_CUDA_in_bytes, hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_r_init, r_init.data(), size_of_r_init_in_bytes, hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_Dn_IN_F, Dn_IN_F.data(), size_of_Dn_IN_F_in_bytes, hipMemcpyHostToDevice));
    //  Copy the data
    CUDA_CHECK(hipMemcpy(d_Dn_NN_inv, Dn_NN_inv.data(), size_of_Dn_NN_inv_in_bytes, hipMemcpyHostToDevice));

    //  BENCHMARK
    for (auto _ : t_state) {
        auto start = std::chrono::high_resolution_clock::now();
        // TO BENCHMARK: START
        // Launch the kernel: computeIvpKernel
        computeIvpKernel<<<1, number_of_Chebyshev_points-1>>>(d_Dn_IN_F, d_r_init, d_ivp);

        // Launch the kernel: updatePositionbKernel
        updatePositionbKernel<<<1, number_of_Chebyshev_points-1>>>(d_Q_stack_CUDA, d_b, d_ivp);

        // here we used d_b = d_res --> the res values are contained in d_b

        // Compute r_stack = Dn_NN_inv*res
        double alpha_cublas = 1.0;
        double beta_cublas = 0.0;
        CUBLAS_CHECK(hipblasDgemm(cublasH, HIPBLAS_OP_N, HIPBLAS_OP_N, rows_Dn_NN_inv, cols_res, cols_Dn_NN_inv, &alpha_cublas, d_Dn_NN_inv, ld_Dn_NN_inv, d_b, ld_res, &beta_cublas, d_r_stack, ld_r_stack)); //d_b = d_res
        // TO BENCHMARK: END
        auto end = std::chrono::high_resolution_clock::now();
        auto elapsed_seconds = std::chrono::duration_cast<std::chrono::duration<double>>(end - start);

        t_state.SetIterationTime(elapsed_seconds.count());
    }

    //FREEING MEMORY
    CUDA_CHECK(hipMemcpy(r_stack_CUDA.data(), d_r_stack, size_of_r_stack_in_bytes, hipMemcpyDeviceToHost));

    //FREEING MEMORY
    CUDA_CHECK(hipFree(d_Q_stack_CUDA));
    CUDA_CHECK(hipFree(d_b));
    CUDA_CHECK(hipFree(d_r_init));
    CUDA_CHECK(hipFree(d_ivp));
    CUDA_CHECK(hipFree(d_Dn_IN_F));
    CUDA_CHECK(hipFree(d_Dn_NN_inv));
    CUDA_CHECK(hipFree(d_r_stack));

    })->Repetitions(20)->Unit(::benchmark::kMicrosecond);

    #pragma region integratePosition
    // Vectors definitions
    Eigen::Vector3d r_init;
    r_init << 0,
              0,
              0;

    Eigen::MatrixXd Dn_NN_inv = Dn_NN_F.inverse(); //  This matrix remains constant so we can pre invert

    Eigen::Matrix<double, number_of_Chebyshev_points-1, position_dimension> b; //used in updatePositionbKernel
    
    Eigen::MatrixXd ivp(number_of_Chebyshev_points-1, position_dimension);

    Eigen::MatrixXd res(number_of_Chebyshev_points-1, position_dimension);

    // Dimension definition
    const int rows_Dn_NN_inv = Dn_NN_inv.rows();
    const int cols_Dn_NN_inv = Dn_NN_inv.cols();
    const int ld_Dn_NN_inv = rows_Dn_NN_inv;  

    const int rows_res = res.rows();
    const int cols_res = res.cols();
    const int ld_res = rows_res;

    const int rows_r_stack = rows_Dn_NN_inv;
    const int cols_r_stack = cols_res;
    const int ld_r_stack = rows_r_stack;

    Eigen::MatrixXd r_stack_CUDA(rows_r_stack, cols_r_stack); // What we want to calculate 

    // Compute the memory occupation for computeIvpKernel/updatePositionbKernel
    const auto size_of_Q_stack_CUDA_in_bytes = t_Q_stack_CUDA.size() * size_of_double;
    const auto size_of_b_in_bytes = b.size() * size_of_double;
    const auto size_of_r_init_in_bytes = r_init.size() * size_of_double;
    const auto size_of_ivp_in_bytes = ivp.size() * size_of_double;
    const auto size_of_Dn_IN_F_in_bytes = Dn_IN_F.size() * size_of_double;
    // Compute the memory occupation
    const auto size_of_Dn_NN_inv_in_bytes = size_of_double * Dn_NN_inv.size();
    const auto size_of_r_stack_in_bytes = size_of_double * rows_r_stack * cols_r_stack;

    // Create Pointers for computeIvpKernel/updatePositionbKernel
    double* d_Q_stack_CUDA;
    double* d_b;
    double* d_r_init;
    double* d_ivp;
    double* d_Dn_IN_F;
    // Create Pointers
    double* d_Dn_NN_inv = nullptr;
    double* d_r_stack = nullptr;

    // Allocate the memory for computeIvpKernel/updatePositionbKernel
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_Q_stack_CUDA), size_of_Q_stack_CUDA_in_bytes));
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_b), size_of_b_in_bytes));
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_r_init), size_of_r_init_in_bytes));
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_ivp), size_of_ivp_in_bytes));
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_Dn_IN_F), size_of_Dn_IN_F_in_bytes));
    // Allocate the memory
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_Dn_NN_inv), size_of_Dn_NN_inv_in_bytes));
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_r_stack), size_of_r_stack_in_bytes));

    //  Copy the data for computeIvpKernel/updatePositionbKernel
    CUDA_CHECK(hipMemcpy(d_Q_stack_CUDA, t_Q_stack_CUDA.data(), size_of_Q_stack_CUDA_in_bytes, hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_r_init, r_init.data(), size_of_r_init_in_bytes, hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_Dn_IN_F, Dn_IN_F.data(), size_of_Dn_IN_F_in_bytes, hipMemcpyHostToDevice));
    //  Copy the data
    CUDA_CHECK(hipMemcpy(d_Dn_NN_inv, Dn_NN_inv.data(), size_of_Dn_NN_inv_in_bytes, hipMemcpyHostToDevice));

    // Launch the kernel: computeIvpKernel
    computeIvpKernel<<<1, number_of_Chebyshev_points-1>>>(d_Dn_IN_F, d_r_init, d_ivp);

    // Launch the kernel: updatePositionbKernel
    updatePositionbKernel<<<1, number_of_Chebyshev_points-1>>>(d_Q_stack_CUDA, d_b, d_ivp);

    // here we used d_b = d_res --> the res values are contained in d_b

    // Compute r_stack = Dn_NN_inv*res
    double alpha_cublas = 1.0;
    double beta_cublas = 0.0;
    CUBLAS_CHECK(hipblasDgemm(cublasH, HIPBLAS_OP_N, HIPBLAS_OP_N, rows_Dn_NN_inv, cols_res, cols_Dn_NN_inv, &alpha_cublas, d_Dn_NN_inv, ld_Dn_NN_inv, d_b, ld_res, &beta_cublas, d_r_stack, ld_r_stack)); //d_b = d_res
    //FREEING MEMORY
    CUDA_CHECK(hipMemcpy(r_stack_CUDA.data(), d_r_stack, size_of_r_stack_in_bytes, hipMemcpyDeviceToHost));

    //FREEING MEMORY
    CUDA_CHECK(hipFree(d_Q_stack_CUDA));
    CUDA_CHECK(hipFree(d_b));
    CUDA_CHECK(hipFree(d_r_init));
    CUDA_CHECK(hipFree(d_ivp));
    CUDA_CHECK(hipFree(d_Dn_IN_F));
    CUDA_CHECK(hipFree(d_Dn_NN_inv));
    CUDA_CHECK(hipFree(d_r_stack));

    #pragma endregion integratePosition

    return r_stack_CUDA;
}




// Used to build Lambda_stack:
__global__ void updateCMatrixKernel(const double* d_K_stack, const double* D_NN, double* C_NN) {

    int i = threadIdx.x;

    // Transpose of the K_hat matrix is equivalent to the K_hat matrix itself multiplied by -1

    // v_hat <<  0   ,  -t_v(2),   t_v(1),
    //         t_v(2),     0   ,  -t_v(0),
    //        -t_v(1),   t_v(0),     0   ;

    // v_hat TRANSPOSE <<  0   ,  t_v(2),   -t_v(1),
    //                  -t_v(2),     0   ,   t_v(0),
    //                  t_v(1),   -t_v(0),     0   ;

    #pragma region Compute_C_NN
    if (i < number_of_Chebyshev_points-1) {
    int row = 0;
    int col = 1;
    int row_index = row * (number_of_Chebyshev_points - 1) + i;
    int col_index = col * (number_of_Chebyshev_points - 1) + i;
    C_NN[row_index * quaternion_state_dimension + col_index] = D_NN[row_index * quaternion_state_dimension + col_index] - d_K_stack[3*i+2];


    row = 0;
    col = 2;
    row_index = row * (number_of_Chebyshev_points - 1) + i;
    col_index = col * (number_of_Chebyshev_points - 1) + i;
    C_NN[row_index * quaternion_state_dimension + col_index] = D_NN[row_index * quaternion_state_dimension + col_index] + d_K_stack[3*i+1];


    row = 1;
    col = 0;
    row_index = row * (number_of_Chebyshev_points - 1) + i;
    col_index = col * (number_of_Chebyshev_points - 1) + i;
    C_NN[row_index * quaternion_state_dimension + col_index] = D_NN[row_index * quaternion_state_dimension + col_index] + d_K_stack[3*i+2];

    row = 1;
    col = 2;
    row_index = row * (number_of_Chebyshev_points - 1) + i;
    col_index = col * (number_of_Chebyshev_points - 1) + i;
    C_NN[row_index * quaternion_state_dimension + col_index] = D_NN[row_index * quaternion_state_dimension + col_index] - d_K_stack[3*i+0];

    row = 2;
    col = 0;
    row_index = row * (number_of_Chebyshev_points - 1) + i;
    col_index = col * (number_of_Chebyshev_points - 1) + i;
    C_NN[row_index * quaternion_state_dimension + col_index] = D_NN[row_index * quaternion_state_dimension + col_index] - d_K_stack[3*i+1];

    row = 2;
    col = 1;
    row_index = row * (number_of_Chebyshev_points - 1) + i;
    col_index = col * (number_of_Chebyshev_points - 1) + i;
    C_NN[row_index * quaternion_state_dimension + col_index] = D_NN[row_index * quaternion_state_dimension + col_index] + d_K_stack[3*i+0];
    }
    #pragma endregion
}

__global__ void computeNbarKernel(const double* t_Q_stack_CUDA_data, double* Nbar_stack_data) {
int i = threadIdx.x;

    const double g = 9.81; // m/s^2
    const double radius = 0.001; // m
    const double A = M_PI * radius * radius;
    const double rho = 7800; // kg/m^3

    Eigen::Map<const Eigen::VectorXd> t_Q_stack_CUDA(t_Q_stack_CUDA_data, (number_of_Chebyshev_points - 1) * 4);
    Eigen::Map<Eigen::VectorXd> Nbar_stack(Nbar_stack_data, (number_of_Chebyshev_points - 1) * 3);

    if (i < number_of_Chebyshev_points - 1) {
        Eigen::Quaterniond Qbar(t_Q_stack_CUDA(i), t_Q_stack_CUDA(i + (number_of_Chebyshev_points - 1)),
                                t_Q_stack_CUDA(i + 2 * (number_of_Chebyshev_points - 1)),
                                t_Q_stack_CUDA(i + 3 * (number_of_Chebyshev_points - 1)));

        double R[9];
        quaternionToRotationMatrix(Qbar.coeffs().data(), R); //Qbar.coeffs().data() returns a pointer to the raw data of the quaternion coefficients of Qbar

        double Fg = -A * g * rho;
        double Nbar[3] = {  R[6]*Fg, 
                            R[7]*Fg,
                            R[8]*Fg
        };

        Nbar_stack(i) = Nbar[0];
        Nbar_stack(i + (number_of_Chebyshev_points - 1)) = Nbar[1];
        Nbar_stack(i + 2 * (number_of_Chebyshev_points - 1)) = Nbar[2];
    }
}

Eigen::MatrixXd integrateInternalForces(Eigen::MatrixXd t_Q_stack_CUDA)
{   
    ::benchmark::RegisterBenchmark("Integrate Internal Forces:", [&](::benchmark::State &t_state){

    t_state.counters = {
        {"na", na},
        {"ne", ne},
        {"Cheb pts", number_of_Chebyshev_points}
    };

    // INITIALISATION
    // Vectors definitions
    const Eigen::MatrixXd D_NN = Eigen::KroneckerProduct<Eigen::MatrixXd,Eigen::MatrixXd>(Eigen::MatrixXd::Identity(lambda_dimension/2, lambda_dimension/2), Dn_NN_B);
    const Eigen::MatrixXd D_IN = Eigen::KroneckerProduct<Eigen::MatrixXd,Eigen::MatrixXd>(Eigen::MatrixXd::Identity(lambda_dimension/2, lambda_dimension/2), Dn_IN_B);

    Eigen::MatrixXd C_NN = D_NN;
    
    Eigen::VectorXd N_init(lambda_dimension/2);
    N_init << 1, 0, 0;

    Eigen::MatrixXd beta = Eigen::MatrixXd::Zero((lambda_dimension/2)*(number_of_Chebyshev_points-1),1);

    // Dimension definition
    const int rows_C_NN = C_NN.rows();
    const int cols_C_NN = C_NN.cols();
    const int ld_C_NN = rows_C_NN;

    const int rows_D_IN = D_IN.rows();
    const int cols_D_IN = D_IN.cols();
    const int ld_D_IN = rows_D_IN;

    const int rows_N_init = N_init.rows();
    const int cols_N_init = N_init.cols();
    const int ld_N_init = rows_N_init;

    const int rows_beta = beta.rows();
    const int cols_beta = beta.cols();
    const int ld_beta = rows_beta;

    const int rows_N_stack = rows_beta;
    const int cols_N_stack = cols_beta;

    Eigen::MatrixXd N_stack_CUDA(rows_N_stack, cols_N_stack); //What we want to calculate
    
    int info = 0;
    int lwork = 0;

    // Compute the memory occupation for updateCMatrixKernel
    const auto size_of_D_NN_in_bytes = D_NN.size() * size_of_double;
    const auto size_of_Q_stack_CUDA_in_bytes = size_of_double * t_Q_stack_CUDA.size();
    const auto size_of_Nbar_stack_in_bytes = size_of_double * beta.size(); // Same dimension of beta (beta = -Nbar)
    // Compute the memory occupation
    const auto size_of_C_NN_in_bytes = size_of_double * C_NN.size();
    const auto size_of_D_IN_in_bytes = size_of_double * D_IN.size();
    const auto size_of_N_init_in_bytes = size_of_double * N_init.size();
    const auto size_of_beta_in_bytes = size_of_double * beta.size();
    const auto size_of_N_stack_in_bytes = size_of_double * rows_N_stack * cols_N_stack;
    
    // Create Pointers for updateCMatrixKernel/computeNbarKernel
    double* d_D_NN = nullptr;
    double* d_C_NN = nullptr;
    double* d_Q_stack_CUDA = nullptr;
    double* d_Nbar_stack = nullptr;
    // Create Pointers
    double* d_D_IN = nullptr;
    double* d_N_init = nullptr;
    double* d_beta = nullptr;
    double* d_N_stack = nullptr;
    double* d_work = nullptr;
    int* d_info = nullptr;

    // Allocate the memory for updateCMatrixKernel/computeNbarKernel
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_qe), size_of_qe_in_bytes));
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_D_NN), size_of_D_NN_in_bytes));
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_C_NN), size_of_D_NN_in_bytes));
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_Q_stack_CUDA), size_of_Q_stack_CUDA_in_bytes));
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_Nbar_stack), size_of_Nbar_stack_in_bytes));
    // Allocate the memory
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_D_IN), size_of_D_IN_in_bytes));
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_N_init), size_of_N_init_in_bytes));
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_beta), size_of_beta_in_bytes));
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_N_stack), size_of_N_stack_in_bytes));
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_info), sizeof(int)));

    //  Copy the data for updateCMatrixKernel/computeNbarKernel
    CUDA_CHECK(hipMemcpy(d_D_NN, D_NN.data(), size_of_D_NN_in_bytes, hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_C_NN, C_NN.data(), size_of_D_NN_in_bytes, hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_Q_stack_CUDA, t_Q_stack_CUDA.data(), size_of_Q_stack_CUDA_in_bytes, hipMemcpyHostToDevice));
    //  Copy the data
    CUDA_CHECK(hipMemcpy(d_D_IN, D_IN.data(), size_of_D_IN_in_bytes, hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_N_init, N_init.data(), size_of_N_init_in_bytes, hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_beta, beta.data(), size_of_beta_in_bytes, hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_info, &info, sizeof(int), hipMemcpyHostToDevice));

    // Allocates buffer size for the LU decomposition
    hipsolverStatus_t status = hipsolverDnDgetrf_bufferSize(cusolverH, cols_C_NN, cols_C_NN, d_C_NN, ld_C_NN, &lwork);
    if (status != HIPSOLVER_STATUS_SUCCESS)
    {
        std::cerr << "cusolver error: " << getCusolverErrorString(status) << std::endl;
    };

    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_work), size_of_double * lwork));

    //  BENCHMARK
    for (auto _ : t_state) {
        auto start = std::chrono::high_resolution_clock::now();
        // TO BENCHMARK: START
        // Launch kernel: updateCMatrixKernel
        updateCMatrixKernel<<<1, number_of_Chebyshev_points>>>(d_K_stack, d_D_NN, d_C_NN);
        
        // Launch the kernel: computeNbarKernel
        computeNbarKernel<<<1, number_of_Chebyshev_points - 1>>>(d_Q_stack_CUDA, d_Nbar_stack);

        // we used d_Nbar_stack instead of d_beta

        // res = -D_IN*N_init - Nbar_stack
        double alpha_cublas = -1.0;
        // IMPORTANT: Normal equation is +beta but since beta is holding Nbar_stack and not -Nbar_stack, we need to change the sign of beta_cublas to compensate
        double beta_cublas = -1.0;
        CUBLAS_CHECK(hipblasDgemm(cublasH, HIPBLAS_OP_N, HIPBLAS_OP_N, rows_D_IN, cols_N_init, cols_D_IN, &alpha_cublas, d_D_IN, ld_D_IN, d_N_init, ld_N_init, &beta_cublas, d_Nbar_stack, ld_beta));

        // LU factorization
        CUSOLVER_CHECK(hipsolverDnDgetrf(cusolverH, cols_C_NN, cols_C_NN, d_C_NN, ld_C_NN, d_work, NULL, d_info));

        // Solving the final system
        CUSOLVER_CHECK(hipsolverDnDgetrs(cusolverH, HIPBLAS_OP_N, cols_C_NN, 1, d_C_NN, ld_C_NN, NULL, d_Nbar_stack, ld_beta, d_info));
        // TO BENCHMARK: END
        auto end = std::chrono::high_resolution_clock::now();
        auto elapsed_seconds = std::chrono::duration_cast<std::chrono::duration<double>>(end - start);

        t_state.SetIterationTime(elapsed_seconds.count());
    }

    CUDA_CHECK(hipMemcpy(N_stack_CUDA.data(), d_Nbar_stack, size_of_beta_in_bytes, hipMemcpyDeviceToHost));

    //FREEING MEMORY
    CUDA_CHECK(hipFree(d_beta));
    CUDA_CHECK(hipFree(d_C_NN));
    CUDA_CHECK(hipFree(d_D_IN));
    CUDA_CHECK(hipFree(d_info));
    CUDA_CHECK(hipFree(d_N_init));
    CUDA_CHECK(hipFree(d_N_stack));
    CUDA_CHECK(hipFree(d_work));
    CUDA_CHECK(hipFree(d_Q_stack_CUDA));
    CUDA_CHECK(hipFree(d_Nbar_stack));
    CUDA_CHECK(hipFree(d_D_NN));

    })->Repetitions(20)->Unit(::benchmark::kMicrosecond);

    #pragma region integrateInternalForces
    // INITIALISATION
    // Vectors definitions
    const Eigen::MatrixXd D_NN = Eigen::KroneckerProduct<Eigen::MatrixXd,Eigen::MatrixXd>(Eigen::MatrixXd::Identity(lambda_dimension/2, lambda_dimension/2), Dn_NN_B);
    const Eigen::MatrixXd D_IN = Eigen::KroneckerProduct<Eigen::MatrixXd,Eigen::MatrixXd>(Eigen::MatrixXd::Identity(lambda_dimension/2, lambda_dimension/2), Dn_IN_B);

    Eigen::MatrixXd C_NN = D_NN;
    
    Eigen::VectorXd N_init(lambda_dimension/2);
    N_init << 1, 0, 0;

    Eigen::MatrixXd beta = Eigen::MatrixXd::Zero((lambda_dimension/2)*(number_of_Chebyshev_points-1),1);

    // Dimension definition
    const int rows_C_NN = C_NN.rows();
    const int cols_C_NN = C_NN.cols();
    const int ld_C_NN = rows_C_NN;

    const int rows_D_IN = D_IN.rows();
    const int cols_D_IN = D_IN.cols();
    const int ld_D_IN = rows_D_IN;

    const int rows_N_init = N_init.rows();
    const int cols_N_init = N_init.cols();
    const int ld_N_init = rows_N_init;

    const int rows_beta = beta.rows();
    const int cols_beta = beta.cols();
    const int ld_beta = rows_beta;

    const int rows_N_stack = rows_beta;
    const int cols_N_stack = cols_beta;

    Eigen::MatrixXd N_stack_CUDA(rows_N_stack, cols_N_stack); //What we want to calculate
    
    int info = 0;
    int lwork = 0;

    // Compute the memory occupation for updateCMatrixKernel
    const auto size_of_D_NN_in_bytes = D_NN.size() * size_of_double;
    const auto size_of_Q_stack_CUDA_in_bytes = size_of_double * t_Q_stack_CUDA.size();
    const auto size_of_Nbar_stack_in_bytes = size_of_double * beta.size(); // Same dimension of beta (beta = -Nbar)
    // Compute the memory occupation
    const auto size_of_C_NN_in_bytes = size_of_double * C_NN.size();
    const auto size_of_D_IN_in_bytes = size_of_double * D_IN.size();
    const auto size_of_N_init_in_bytes = size_of_double * N_init.size();
    const auto size_of_beta_in_bytes = size_of_double * beta.size();
    const auto size_of_N_stack_in_bytes = size_of_double * rows_N_stack * cols_N_stack;
    
    // Create Pointers for updateCMatrixKernel/computeNbarKernel
    double* d_D_NN = nullptr;
    double* d_C_NN = nullptr;
    double* d_Q_stack_CUDA = nullptr;
    double* d_Nbar_stack = nullptr;
    // Create Pointers
    double* d_D_IN = nullptr;
    double* d_N_init = nullptr;
    double* d_beta = nullptr;
    double* d_N_stack = nullptr;
    double* d_work = nullptr;
    int* d_info = nullptr;

    // Allocate the memory for updateCMatrixKernel/computeNbarKernel
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_qe), size_of_qe_in_bytes));
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_D_NN), size_of_D_NN_in_bytes));
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_C_NN), size_of_D_NN_in_bytes));
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_Q_stack_CUDA), size_of_Q_stack_CUDA_in_bytes));
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_Nbar_stack), size_of_Nbar_stack_in_bytes));
    // Allocate the memory
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_D_IN), size_of_D_IN_in_bytes));
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_N_init), size_of_N_init_in_bytes));
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_beta), size_of_beta_in_bytes));
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_N_stack), size_of_N_stack_in_bytes));
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_info), sizeof(int)));

    //  Copy the data for updateCMatrixKernel/computeNbarKernel
    CUDA_CHECK(hipMemcpy(d_D_NN, D_NN.data(), size_of_D_NN_in_bytes, hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_C_NN, C_NN.data(), size_of_D_NN_in_bytes, hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_Q_stack_CUDA, t_Q_stack_CUDA.data(), size_of_Q_stack_CUDA_in_bytes, hipMemcpyHostToDevice));
    //  Copy the data
    CUDA_CHECK(hipMemcpy(d_D_IN, D_IN.data(), size_of_D_IN_in_bytes, hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_N_init, N_init.data(), size_of_N_init_in_bytes, hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_beta, beta.data(), size_of_beta_in_bytes, hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_info, &info, sizeof(int), hipMemcpyHostToDevice));

    // Allocates buffer size for the LU decomposition
    hipsolverStatus_t status = hipsolverDnDgetrf_bufferSize(cusolverH, cols_C_NN, cols_C_NN, d_C_NN, ld_C_NN, &lwork);
    if (status != HIPSOLVER_STATUS_SUCCESS)
    {
        std::cerr << "cusolver error: " << getCusolverErrorString(status) << std::endl;
    };

    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_work), size_of_double * lwork));

    // Launch kernel: updateCMatrixKernel
    updateCMatrixKernel<<<1, number_of_Chebyshev_points>>>(d_K_stack, d_D_NN, d_C_NN);
    
    // Launch the kernel: computeNbarKernel
    computeNbarKernel<<<1, number_of_Chebyshev_points - 1>>>(d_Q_stack_CUDA, d_Nbar_stack);

    // we used d_Nbar_stack instead of d_beta

    // res = -D_IN*N_init - Nbar_stack
    double alpha_cublas = -1.0;
    // IMPORTANT: Normal equation is +beta but since beta is holding Nbar_stack and not -Nbar_stack, we need to change the sign of beta_cublas to compensate
    double beta_cublas = -1.0;
    CUBLAS_CHECK(hipblasDgemm(cublasH, HIPBLAS_OP_N, HIPBLAS_OP_N, rows_D_IN, cols_N_init, cols_D_IN, &alpha_cublas, d_D_IN, ld_D_IN, d_N_init, ld_N_init, &beta_cublas, d_Nbar_stack, ld_beta));

    // LU factorization
    CUSOLVER_CHECK(hipsolverDnDgetrf(cusolverH, cols_C_NN, cols_C_NN, d_C_NN, ld_C_NN, d_work, NULL, d_info));

    // Solving the final system
    CUSOLVER_CHECK(hipsolverDnDgetrs(cusolverH, HIPBLAS_OP_N, cols_C_NN, 1, d_C_NN, ld_C_NN, NULL, d_Nbar_stack, ld_beta, d_info));


    CUDA_CHECK(hipMemcpy(N_stack_CUDA.data(), d_Nbar_stack, size_of_beta_in_bytes, hipMemcpyDeviceToHost));

    //FREEING MEMORY
    CUDA_CHECK(hipFree(d_beta));
    CUDA_CHECK(hipFree(d_C_NN));
    CUDA_CHECK(hipFree(d_D_IN));
    CUDA_CHECK(hipFree(d_info));
    CUDA_CHECK(hipFree(d_N_init));
    CUDA_CHECK(hipFree(d_N_stack));
    CUDA_CHECK(hipFree(d_work));
    CUDA_CHECK(hipFree(d_Q_stack_CUDA));
    CUDA_CHECK(hipFree(d_Nbar_stack));
    CUDA_CHECK(hipFree(d_D_NN));

    #pragma endregion integrateInternalForces

    return N_stack_CUDA;
}

__global__ void updateCouplesbKernel(const double* t_N_stack_CUDA, double* d_beta) {
    int idx = threadIdx.x;

    const Eigen::Vector3d C_bar = Eigen::Vector3d::Zero();
    Eigen::Vector3d N;

    if (idx < number_of_Chebyshev_points - 1) {
        Eigen::VectorXd Gamma(lambda_dimension / 2);
        Gamma << 1, 0, 0;

        // Construct the skew-symmetric matrix manually
        Eigen::Matrix3d skewGamma;
        skewGamma << 0, -Gamma(2), Gamma(1),
                     Gamma(2), 0, -Gamma(0),
                    -Gamma(1), Gamma(0), 0;

        int offset = idx * lambda_dimension / 2;

        for (int i = 0; i < lambda_dimension / 2; ++i) {
            N(i) = t_N_stack_CUDA[offset + i];
        }

        // Perform b = skewGamma.transpose() * N - C_bar
        double b[3] = { skewGamma(2)*N(1)-skewGamma(1)*N(2)-C_bar(0),
                            -skewGamma(2)*N(0)+skewGamma(0)*N(2)-C_bar(1),
                            skewGamma(1)*N(0)-skewGamma(0)*N(1)-C_bar(2)};


        for (int i = 0; i < lambda_dimension / 2; ++i) {
            d_beta[offset + i] = b[i];
        }
    }
}

Eigen::MatrixXd integrateInternalCouples(Eigen::MatrixXd t_N_stack_CUDA)
{    
    ::benchmark::RegisterBenchmark("Integrate Internal Couples:", [&](::benchmark::State &t_state){

    t_state.counters = {
        {"na", na},
        {"ne", ne},
        {"Cheb pts", number_of_Chebyshev_points}
    };

    // INITIALISATION
    // Vectors definitions
    const Eigen::MatrixXd D_NN = Eigen::KroneckerProduct<Eigen::MatrixXd,Eigen::MatrixXd>(Eigen::MatrixXd::Identity(lambda_dimension/2, lambda_dimension/2), Dn_NN_B);
    const Eigen::MatrixXd D_IN = Eigen::KroneckerProduct<Eigen::MatrixXd,Eigen::MatrixXd>(Eigen::MatrixXd::Identity(lambda_dimension/2, lambda_dimension/2), Dn_IN_B);

    Eigen::MatrixXd C_NN = D_NN;

    Eigen::MatrixXd beta_NN = Eigen::MatrixXd::Zero((lambda_dimension/2)*(number_of_Chebyshev_points-1), 1);
    
    Eigen::VectorXd C_init(lambda_dimension/2);
    C_init << 1, 0, 0;
    
    Eigen::MatrixXd C_stack_CUDA(t_N_stack_CUDA.rows(), t_N_stack_CUDA.cols()); //What we want to calculate

    // Dimension definition
    const int rows_C_NN = C_NN.rows();
    const int cols_C_NN = C_NN.cols();
    const int ld_C_NN = rows_C_NN;

    const int rows_D_IN = D_IN.rows();
    const int cols_D_IN = D_IN.cols();
    const int ld_D_IN = rows_D_IN;

    const int rows_C_init = C_init.rows();
    const int cols_C_init = C_init.cols();
    const int ld_C_init = rows_C_init;

    const int rows_beta_NN = beta_NN.rows();
    const int cols_beta_NN = beta_NN.cols();
    const int ld_beta_NN = rows_beta_NN;
    
    int info = 0;
    int lwork = 0;

    // Compute the memory occupation for updateCMatrixKernel/computeNbarKernel
    const auto size_of_D_NN_in_bytes = D_NN.size() * size_of_double;
    const auto size_of_N_stack_CUDA_in_bytes = size_of_double * t_N_stack_CUDA.size();
    const auto size_of_Nbar_stack_in_bytes = size_of_double * beta_NN.size(); // Same dimension of beta (beta = -Nbar)
    const auto size_of_beta_NN_in_bytes = size_of_double * beta_NN.size();
    // Compute the memory occupation
    const auto size_of_C_NN_in_bytes = size_of_double * C_NN.size();
    const auto size_of_D_IN_in_bytes = size_of_double * D_IN.size();
    const auto size_of_C_init_in_bytes = size_of_double * C_init.size();
    const auto size_of_N_stack_in_bytes = size_of_double * t_N_stack_CUDA.size();
    
    // Create Pointers for updateCMatrixKernel/computeNbarKernel
    double* d_D_NN = nullptr;
    double* d_C_NN = nullptr;
    double* d_N_stack_CUDA = nullptr;
    double* d_beta_NN = nullptr;
    // Create Pointers
    double* d_D_IN = nullptr;
    double* d_C_init = nullptr;
    double* d_work = nullptr;
    int* d_info = nullptr;

    // Allocate the memory for updateCMatrixKernel/computeNbarKernel
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_D_NN), size_of_D_NN_in_bytes));
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_C_NN), size_of_D_NN_in_bytes));
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_N_stack_CUDA), size_of_N_stack_CUDA_in_bytes));
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_beta_NN), size_of_beta_NN_in_bytes));
    // Allocate the memory
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_D_IN), size_of_D_IN_in_bytes));
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_C_init), size_of_C_init_in_bytes));
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_info), sizeof(int)));

    //  Copy the data for updateCMatrixKernel/computeNbarKernel
    CUDA_CHECK(hipMemcpy(d_D_NN, D_NN.data(), size_of_D_NN_in_bytes, hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_C_NN, C_NN.data(), size_of_D_NN_in_bytes, hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_N_stack_CUDA, t_N_stack_CUDA.data(), size_of_N_stack_CUDA_in_bytes, hipMemcpyHostToDevice));
    //  Copy the data
    CUDA_CHECK(hipMemcpy(d_D_IN, D_IN.data(), size_of_D_IN_in_bytes, hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_C_init, C_init.data(), size_of_C_init_in_bytes, hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_info, &info, sizeof(int), hipMemcpyHostToDevice));

    // Allocates buffer size for the LU decomposition
    hipsolverStatus_t status = hipsolverDnDgetrf_bufferSize(cusolverH, rows_C_NN, cols_C_NN, d_C_NN, ld_C_NN, &lwork);
    if (status != HIPSOLVER_STATUS_SUCCESS)
    {
        std::cerr << "cusolver error: " << getCusolverErrorString(status) << std::endl;
    };

    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_work), size_of_double * lwork));

    //  BENCHMARK
    for (auto _ : t_state) {
        auto start = std::chrono::high_resolution_clock::now();
        // TO BENCHMARK: START
        // Launch the kernel: updateCMatrixKernel
        updateCMatrixKernel<<<1, number_of_Chebyshev_points-1>>>(d_K_stack, d_D_NN, d_C_NN);

        // Launch the kernel: computeNbarKernel
        updateCouplesbKernel<<<1, number_of_Chebyshev_points - 1>>>(d_N_stack_CUDA, d_beta_NN);

        double alpha_cublas = -1.0;
        double beta_cublas = 1.0;
        // res = -D_IN*C_init + beta_NN
        CUBLAS_CHECK(hipblasDgemm(cublasH, HIPBLAS_OP_N, HIPBLAS_OP_N, rows_D_IN, cols_C_init, cols_D_IN, &alpha_cublas, d_D_IN, ld_D_IN, d_C_init, ld_C_init, &beta_cublas, d_beta_NN, ld_beta_NN));

        // LU factorization
        CUSOLVER_CHECK(hipsolverDnDgetrf(cusolverH, rows_C_NN, cols_C_NN, d_C_NN, ld_C_NN, d_work, NULL, d_info));

        // Solving the final system
        CUSOLVER_CHECK(hipsolverDnDgetrs(cusolverH, HIPBLAS_OP_N, rows_C_NN, 1, d_C_NN, ld_C_NN, NULL, d_beta_NN, ld_beta_NN, d_info));
        // TO BENCHMARK: END
        auto end = std::chrono::high_resolution_clock::now();
        auto elapsed_seconds = std::chrono::duration_cast<std::chrono::duration<double>>(end - start);

        t_state.SetIterationTime(elapsed_seconds.count());
    }

    CUDA_CHECK(hipMemcpy(C_stack_CUDA.data(), d_beta_NN, size_of_beta_NN_in_bytes, hipMemcpyDeviceToHost));

    //FREEING MEMORY
    CUDA_CHECK(hipFree(d_beta_NN));
    CUDA_CHECK(hipFree(d_C_init));
    CUDA_CHECK(hipFree(d_C_NN));
    CUDA_CHECK(hipFree(d_D_IN));
    CUDA_CHECK(hipFree(d_N_stack_CUDA));
    CUDA_CHECK(hipFree(d_D_NN));
    CUDA_CHECK(hipFree(d_info));
    CUDA_CHECK(hipFree(d_work));

    })->Repetitions(20)->Unit(::benchmark::kMicrosecond);

    #pragma region integrateInternalCouples
    // INITIALISATION
    // Vectors definitions
    const Eigen::MatrixXd D_NN = Eigen::KroneckerProduct<Eigen::MatrixXd,Eigen::MatrixXd>(Eigen::MatrixXd::Identity(lambda_dimension/2, lambda_dimension/2), Dn_NN_B);
    const Eigen::MatrixXd D_IN = Eigen::KroneckerProduct<Eigen::MatrixXd,Eigen::MatrixXd>(Eigen::MatrixXd::Identity(lambda_dimension/2, lambda_dimension/2), Dn_IN_B);

    Eigen::MatrixXd C_NN = D_NN;

    Eigen::MatrixXd beta_NN = Eigen::MatrixXd::Zero((lambda_dimension/2)*(number_of_Chebyshev_points-1), 1);
    
    Eigen::VectorXd C_init(lambda_dimension/2);
    C_init << 1, 0, 0;
    
    Eigen::MatrixXd C_stack_CUDA(t_N_stack_CUDA.rows(), t_N_stack_CUDA.cols()); //What we want to calculate

    // Dimension definition
    const int rows_C_NN = C_NN.rows();
    const int cols_C_NN = C_NN.cols();
    const int ld_C_NN = rows_C_NN;

    const int rows_D_IN = D_IN.rows();
    const int cols_D_IN = D_IN.cols();
    const int ld_D_IN = rows_D_IN;

    const int rows_C_init = C_init.rows();
    const int cols_C_init = C_init.cols();
    const int ld_C_init = rows_C_init;

    const int rows_beta_NN = beta_NN.rows();
    const int cols_beta_NN = beta_NN.cols();
    const int ld_beta_NN = rows_beta_NN;
    
    int info = 0;
    int lwork = 0;

    // Compute the memory occupation for updateCMatrixKernel/computeNbarKernel
    const auto size_of_D_NN_in_bytes = D_NN.size() * size_of_double;
    const auto size_of_N_stack_CUDA_in_bytes = size_of_double * t_N_stack_CUDA.size();
    const auto size_of_Nbar_stack_in_bytes = size_of_double * beta_NN.size(); // Same dimension of beta (beta = -Nbar)
    const auto size_of_beta_NN_in_bytes = size_of_double * beta_NN.size();
    // Compute the memory occupation
    const auto size_of_C_NN_in_bytes = size_of_double * C_NN.size();
    const auto size_of_D_IN_in_bytes = size_of_double * D_IN.size();
    const auto size_of_C_init_in_bytes = size_of_double * C_init.size();
    const auto size_of_N_stack_in_bytes = size_of_double * t_N_stack_CUDA.size();
    
    // Create Pointers for updateCMatrixKernel/computeNbarKernel
    double* d_D_NN = nullptr;
    double* d_C_NN = nullptr;
    double* d_N_stack_CUDA = nullptr;
    double* d_beta_NN = nullptr;
    // Create Pointers
    double* d_D_IN = nullptr;
    double* d_C_init = nullptr;
    double* d_work = nullptr;
    int* d_info = nullptr;

    // Allocate the memory for updateCMatrixKernel/computeNbarKernel
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_D_NN), size_of_D_NN_in_bytes));
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_C_NN), size_of_D_NN_in_bytes));
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_N_stack_CUDA), size_of_N_stack_CUDA_in_bytes));
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_beta_NN), size_of_beta_NN_in_bytes));
    // Allocate the memory
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_D_IN), size_of_D_IN_in_bytes));
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_C_init), size_of_C_init_in_bytes));
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_info), sizeof(int)));

    //  Copy the data for updateCMatrixKernel/computeNbarKernel
    CUDA_CHECK(hipMemcpy(d_D_NN, D_NN.data(), size_of_D_NN_in_bytes, hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_C_NN, C_NN.data(), size_of_D_NN_in_bytes, hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_N_stack_CUDA, t_N_stack_CUDA.data(), size_of_N_stack_CUDA_in_bytes, hipMemcpyHostToDevice));
    //  Copy the data
    CUDA_CHECK(hipMemcpy(d_D_IN, D_IN.data(), size_of_D_IN_in_bytes, hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_C_init, C_init.data(), size_of_C_init_in_bytes, hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_info, &info, sizeof(int), hipMemcpyHostToDevice));

    // Allocates buffer size for the LU decomposition
    hipsolverStatus_t status = hipsolverDnDgetrf_bufferSize(cusolverH, rows_C_NN, cols_C_NN, d_C_NN, ld_C_NN, &lwork);
    if (status != HIPSOLVER_STATUS_SUCCESS)
    {
        std::cerr << "cusolver error: " << getCusolverErrorString(status) << std::endl;
    };

    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_work), size_of_double * lwork));


    // Launch the kernel: updateCMatrixKernel
    updateCMatrixKernel<<<1, number_of_Chebyshev_points-1>>>(d_K_stack, d_D_NN, d_C_NN);

    // Launch the kernel: computeNbarKernel
    updateCouplesbKernel<<<1, number_of_Chebyshev_points - 1>>>(d_N_stack_CUDA, d_beta_NN);

    double alpha_cublas = -1.0;
    double beta_cublas = 1.0;
    // res = -D_IN*C_init + beta_NN
    CUBLAS_CHECK(hipblasDgemm(cublasH, HIPBLAS_OP_N, HIPBLAS_OP_N, rows_D_IN, cols_C_init, cols_D_IN, &alpha_cublas, d_D_IN, ld_D_IN, d_C_init, ld_C_init, &beta_cublas, d_beta_NN, ld_beta_NN));

    // LU factorization
    CUSOLVER_CHECK(hipsolverDnDgetrf(cusolverH, rows_C_NN, cols_C_NN, d_C_NN, ld_C_NN, d_work, NULL, d_info));

    // Solving the final system
    CUSOLVER_CHECK(hipsolverDnDgetrs(cusolverH, HIPBLAS_OP_N, rows_C_NN, 1, d_C_NN, ld_C_NN, NULL, d_beta_NN, ld_beta_NN, d_info));

    CUDA_CHECK(hipMemcpy(C_stack_CUDA.data(), d_beta_NN, size_of_beta_NN_in_bytes, hipMemcpyDeviceToHost));

    //FREEING MEMORY
    CUDA_CHECK(hipFree(d_beta_NN));
    CUDA_CHECK(hipFree(d_C_init));
    CUDA_CHECK(hipFree(d_C_NN));
    CUDA_CHECK(hipFree(d_D_IN));
    CUDA_CHECK(hipFree(d_N_stack_CUDA));
    CUDA_CHECK(hipFree(d_D_NN));
    CUDA_CHECK(hipFree(d_info));
    CUDA_CHECK(hipFree(d_work));

    #pragma endregion integrateInternalCouples

    return C_stack_CUDA;
}

Eigen::MatrixXd buildLambda(Eigen::MatrixXd t_C_stack_CUDA, Eigen::MatrixXd t_N_stack_CUDA)
{
    Eigen::Vector3d C;
    Eigen::Vector3d N;

    Eigen::VectorXd lambda(lambda_dimension);

    Eigen::MatrixXd Lambda_stack(lambda_dimension*(number_of_Chebyshev_points-1), 1);

    for (unsigned int i = 0; i < number_of_Chebyshev_points-1; ++i) {

        N << t_N_stack_CUDA(i),
             t_N_stack_CUDA(i  +  (number_of_Chebyshev_points-1)),
             t_N_stack_CUDA(i + 2*(number_of_Chebyshev_points-1));

        C << t_C_stack_CUDA(i),
             t_C_stack_CUDA(i  +  (number_of_Chebyshev_points-1)),
             t_C_stack_CUDA(i + 2*(number_of_Chebyshev_points-1));

        lambda << C, N;

        Lambda_stack.block<6,1>(i*lambda_dimension,0) = lambda;
    }

    return Lambda_stack;
}





// Used to build Qa_stack
// CUDA kernel function to update Qad_vector_b
__global__ void updateQad_vector_bKernel(double* d_Lambda_stack, double* B_NN, double* d_Phi_stack, int Qa_dimension) {
    int tid = threadIdx.x;

    if (tid < number_of_Chebyshev_points - 1) {
        // Create Eigen objects for B_NN and b
        Eigen::Map<Eigen::MatrixXd> B_NN_mat(B_NN, number_of_Chebyshev_points - 1, Qa_dimension);
        Eigen::VectorXd b(Qa_dimension);

        // // Create Eigen object for B
        // Eigen::Matrix<double, 6, 3> B;
        // B.block(0, 0, 3, 3).setIdentity();
        // B.block(3, 0, 3, 3).setZero();

        // The B mtrix is not used here because it contains only ones and zeros and it has been already taken into account
        b(0) = d_Phi_stack[0+tid]*d_Lambda_stack[0+tid] + d_Phi_stack[6+tid]*d_Lambda_stack[1+tid] + d_Phi_stack[12+tid]*d_Lambda_stack[2+tid];
        b(1) = d_Phi_stack[1+tid]*d_Lambda_stack[0+tid] + d_Phi_stack[7+tid]*d_Lambda_stack[1+tid] + d_Phi_stack[13+tid]*d_Lambda_stack[2+tid];
        b(2) = d_Phi_stack[2+tid]*d_Lambda_stack[0+tid] + d_Phi_stack[8+tid]*d_Lambda_stack[1+tid] + d_Phi_stack[14+tid]*d_Lambda_stack[2+tid];
        b(3) = d_Phi_stack[3+tid]*d_Lambda_stack[0+tid] + d_Phi_stack[9+tid]*d_Lambda_stack[1+tid] + d_Phi_stack[15+tid]*d_Lambda_stack[2+tid];
        b(4) = d_Phi_stack[4+tid]*d_Lambda_stack[0+tid] + d_Phi_stack[10+tid]*d_Lambda_stack[1+tid] + d_Phi_stack[16+tid]*d_Lambda_stack[2+tid];
        b(5) = d_Phi_stack[5+tid]*d_Lambda_stack[0+tid] + d_Phi_stack[11+tid]*d_Lambda_stack[1+tid] + d_Phi_stack[17+tid]*d_Lambda_stack[2+tid];
        
        // Set the computed b in the B_NN matrix
        B_NN_mat.row(tid) = b.transpose();

    }
}

Eigen::MatrixXd integrateGeneralisedForces(Eigen::MatrixXd t_Lambda_stack)
{    
    ::benchmark::RegisterBenchmark("Integrate Generalised Forces:", [&](::benchmark::State &t_state){

    t_state.counters = {
        {"na", na},
        {"ne", ne},
        {"Cheb pts", number_of_Chebyshev_points}
    };

    // INITIALISATION
    // Vectors definitions
    Eigen::Vector3d Qa_init;
    Qa_init << 0,
               0,
               0;

    Eigen::MatrixXd B_NN(number_of_Chebyshev_points-1, Qa_dimension);

    Eigen::MatrixXd Dn_NN_inv = Dn_NN_B.inverse(); // Dn_NN is constant so we can pre-invert

    //Definition of matrices dimensions.
    const int rows_B_NN = B_NN.rows();
    const int cols_B_NN = B_NN.cols();
    const int ld_B_NN = rows_B_NN;

    const int rows_Dn_NN_inv = Dn_NN_inv.rows();
    const int cols_Dn_NN_inv = Dn_NN_inv.cols();
    const int ld_Dn_NN_inv = rows_Dn_NN_inv;

    const int rows_Qa_stack = rows_Dn_NN_inv;
    const int cols_Qa_stack = cols_B_NN;
    const int ld_Qa_stack = rows_Qa_stack;

    Eigen::MatrixXd Qa_stack_CUDA(rows_Qa_stack, cols_Qa_stack); // What we want to calculate

    // Compute the memory occupation for updateQad_vector_bKernel
    const auto size_of_B_NN_in_bytes = B_NN.size() * size_of_double;
    const auto size_of_Lambda_stack_in_bytes = t_Lambda_stack.size() * size_of_double;
    // Compute the memory occupation
    const auto size_of_Dn_NN_inv_in_bytes = size_of_double * Dn_NN_inv.size();
    const auto size_of_Qa_stack_in_bytes = size_of_double * rows_Qa_stack * cols_Qa_stack;
    
    // Create Pointers for updateQad_vector_bKernel
    double* d_B_NN = nullptr;
    double* d_Lambda_stack = nullptr;
    // Create Pointers
    double* d_Dn_NN_inv = nullptr;    
    double* d_Qa_stack = nullptr;

    // Allocate the memory for updateQad_vector_bKernel
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_B_NN), size_of_B_NN_in_bytes));
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_Lambda_stack), size_of_Lambda_stack_in_bytes));
    // Allocate the memory
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_Dn_NN_inv), size_of_Dn_NN_inv_in_bytes));
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_Qa_stack), size_of_Qa_stack_in_bytes));
    
    //  Copy the data for updateQad_vector_bKernel
    CUDA_CHECK(hipMemcpy(d_Lambda_stack, t_Lambda_stack.data(), size_of_Lambda_stack_in_bytes, hipMemcpyHostToDevice));
    //  Copy the data
    CUDA_CHECK(hipMemcpy(d_Dn_NN_inv, Dn_NN_inv.data(), size_of_Dn_NN_inv_in_bytes, hipMemcpyHostToDevice));

    //  BENCHMARK
    for (auto _ : t_state) {
        auto start = std::chrono::high_resolution_clock::now();
        // TO BENCHMARK: START
        // Launch the kernel: updateQad_vector_bKernel
        updateQad_vector_bKernel<<<1, number_of_Chebyshev_points-1>>>(d_Lambda_stack, d_B_NN, d_Phi_stack, Qa_dimension);

        // Compute Qa_stack = Dn_NN_inv*B_NN
        double alpha_cublas = 1.0;
        double beta_cublas = 0.0;
        CUBLAS_CHECK(hipblasDgemm(cublasH, HIPBLAS_OP_N, HIPBLAS_OP_N, rows_Dn_NN_inv, cols_B_NN, cols_Dn_NN_inv, &alpha_cublas, d_Dn_NN_inv, ld_Dn_NN_inv, d_B_NN, ld_B_NN, &beta_cublas, d_Qa_stack, ld_Qa_stack));
        // TO BENCHMARK: END
        auto end = std::chrono::high_resolution_clock::now();
        auto elapsed_seconds = std::chrono::duration_cast<std::chrono::duration<double>>(end - start);

        t_state.SetIterationTime(elapsed_seconds.count());
    }

    CUDA_CHECK(hipMemcpy(Qa_stack_CUDA.data(), d_Qa_stack, size_of_Qa_stack_in_bytes, hipMemcpyDeviceToHost));

    //FREEING MEMORY
    CUDA_CHECK(hipFree(d_B_NN));
    CUDA_CHECK(hipFree(d_Qa_stack));
    CUDA_CHECK(hipFree(d_Dn_NN_inv));
    CUDA_CHECK(hipFree(d_Lambda_stack));
    
    })->Repetitions(20)->Unit(::benchmark::kMicrosecond);

    #pragma region integrateGeneralisedForces
    // INITIALISATION
    // Vectors definitions
    Eigen::Vector3d Qa_init;
    Qa_init << 0,
               0,
               0;

    Eigen::MatrixXd B_NN(number_of_Chebyshev_points-1, Qa_dimension);

    Eigen::MatrixXd Dn_NN_inv = Dn_NN_B.inverse(); // Dn_NN is constant so we can pre-invert

    //Definition of matrices dimensions.
    const int rows_B_NN = B_NN.rows();
    const int cols_B_NN = B_NN.cols();
    const int ld_B_NN = rows_B_NN;

    const int rows_Dn_NN_inv = Dn_NN_inv.rows();
    const int cols_Dn_NN_inv = Dn_NN_inv.cols();
    const int ld_Dn_NN_inv = rows_Dn_NN_inv;

    const int rows_Qa_stack = rows_Dn_NN_inv;
    const int cols_Qa_stack = cols_B_NN;
    const int ld_Qa_stack = rows_Qa_stack;

    Eigen::MatrixXd Qa_stack_CUDA(rows_Qa_stack, cols_Qa_stack); // What we want to calculate

    // Compute the memory occupation for updateQad_vector_bKernel
    const auto size_of_B_NN_in_bytes = B_NN.size() * size_of_double;
    const auto size_of_Lambda_stack_in_bytes = t_Lambda_stack.size() * size_of_double;
    // Compute the memory occupation
    const auto size_of_Dn_NN_inv_in_bytes = size_of_double * Dn_NN_inv.size();
    const auto size_of_Qa_stack_in_bytes = size_of_double * rows_Qa_stack * cols_Qa_stack;
    
    // Create Pointers for updateQad_vector_bKernel
    double* d_B_NN = nullptr;
    double* d_Lambda_stack = nullptr;
    // Create Pointers
    double* d_Dn_NN_inv = nullptr;    
    double* d_Qa_stack = nullptr;

    // Allocate the memory for updateQad_vector_bKernel
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_B_NN), size_of_B_NN_in_bytes));
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_Lambda_stack), size_of_Lambda_stack_in_bytes));
    // Allocate the memory
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_Dn_NN_inv), size_of_Dn_NN_inv_in_bytes));
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_Qa_stack), size_of_Qa_stack_in_bytes));
    
    //  Copy the data for updateQad_vector_bKernel
    CUDA_CHECK(hipMemcpy(d_Lambda_stack, t_Lambda_stack.data(), size_of_Lambda_stack_in_bytes, hipMemcpyHostToDevice));
    //  Copy the data
    CUDA_CHECK(hipMemcpy(d_Dn_NN_inv, Dn_NN_inv.data(), size_of_Dn_NN_inv_in_bytes, hipMemcpyHostToDevice));

    // Launch the kernel: updateQad_vector_bKernel
    updateQad_vector_bKernel<<<1, number_of_Chebyshev_points-1>>>(d_Lambda_stack, d_B_NN, d_Phi_stack, Qa_dimension);

    // Compute Qa_stack = Dn_NN_inv*B_NN
    double alpha_cublas = 1.0;
    double beta_cublas = 0.0;
    CUBLAS_CHECK(hipblasDgemm(cublasH, HIPBLAS_OP_N, HIPBLAS_OP_N, rows_Dn_NN_inv, cols_B_NN, cols_Dn_NN_inv, &alpha_cublas, d_Dn_NN_inv, ld_Dn_NN_inv, d_B_NN, ld_B_NN, &beta_cublas, d_Qa_stack, ld_Qa_stack));

    CUDA_CHECK(hipMemcpy(Qa_stack_CUDA.data(), d_Qa_stack, size_of_Qa_stack_in_bytes, hipMemcpyDeviceToHost));

    //FREEING MEMORY
    CUDA_CHECK(hipFree(d_B_NN));
    CUDA_CHECK(hipFree(d_Qa_stack));
    CUDA_CHECK(hipFree(d_Dn_NN_inv));
    CUDA_CHECK(hipFree(d_Lambda_stack));
    
    #pragma endregion integrateGeneralisedForces

    return Qa_stack_CUDA;
}


int main(int argc, char *argv[])
{
    //  cuda blas api need CUBLAS_CHECK
    CUBLAS_CHECK(hipblasCreate(&cublasH));
    CUSOLVER_CHECK(hipsolverDnCreate(&cusolverH));

    //  Here we give some value for the strain
    qe <<   0,
            0,
            0,
            1.2877691307032,
           -1.63807499160786,
            0.437406679142598,
            0,
            0,
            0;
    // qe.setZero();
    

    // Here we give some value for the strain rate
    for (unsigned int i = 0; i < number_of_Chebyshev_points; ++i) {
        Phi_stack.block<na, ne>(i*na, 0) =  Phi<na, ne>(Chebyshev_points[i]);
    }

    const auto Q_stack_CUDA = integrateQuaternions();
    // std::cout << "Quaternion Integration : \n" << Q_stack_CUDA << "\n" << std::endl;
    
    const auto r_stack_CUDA = integratePosition(Q_stack_CUDA);
    // std::cout << "Position Integration : \n" << r_stack_CUDA << "\n" << std::endl;

    const auto N_stack_CUDA = integrateInternalForces(Q_stack_CUDA);
    // std::cout << "Internal Forces Integration : \n" << N_stack_CUDA << "\n" << std::endl;

    const auto C_stack_CUDA = integrateInternalCouples(N_stack_CUDA);
    // std::cout << "Internal Couples Integration : \n" << C_stack_CUDA << "\n" << std::endl;
    
    const auto Lambda_stack_CUDA = buildLambda(C_stack_CUDA, N_stack_CUDA);
    //std::cout << "Lambda_stack : \n" << Lambda_stack_CUDA << "\n" << std::endl;

    const auto Qa_stack_CUDA = integrateGeneralisedForces(Lambda_stack_CUDA);
    // std::cout << "Generalized Forces Integration : \n" << Qa_stack_CUDA << std::endl;

    // Benchmark initialization6
    ::benchmark::Initialize(&argc, argv);
    ::benchmark::RunSpecifiedBenchmarks();

    // Destroy the handle
    CUBLAS_CHECK(hipblasDestroy(cublasH));
    CUSOLVER_CHECK(hipsolverDnDestroy(cusolverH));
    CUDA_CHECK(hipDeviceReset());

    return 0;
}